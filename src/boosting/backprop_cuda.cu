#include "hip/hip_runtime.h"
/* backprop_cuda.cc                                                -*- C++ -*-
   Jeremy Barnes, 25 May 2009
   Copyright (c) 2009 Jeremy Barnes.  All rights reserved.

   CUDA-based backprop implementation.
*/

#include "arch/exception.h"
#include "compiler/compiler.h"
#include <cstdio>
#include <iostream>
#include <boost/timer.hpp>
#include <boost/utility.hpp>
#include "arch/cuda/device_data.h"
#include "math/xdiv.h"
#include "perceptron_defs.h"
#include <vector>

using namespace std;


__device__ float transform(float input, int activation)
{
    switch (activation) {
    case ML::ACT_TANH: {
        float pos = __expf(input);
        float neg = __expf(-input);
        return __fdividef(pos + neg, pos - neg);
    }
    case ML::ACT_IDENTITY: return input;
    default:
        return 0.0;
    }
}

/* Given an output and an error, what's the delta? */
__device__ float delta(float output, float error, int activation)
{
    switch (activation) {
    case ML::ACT_TANH:
        return (1.0f - output * output) * error;
    case ML::ACT_IDENTITY: return output * error; 
    default:
        return 0.0;
    }
}

/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    28 multiprocessors with 8 cores each, and so we could compute 28 different
    samples at once).
    
    This kernel will be called with ONE block of threads, with the number
    of threads equal to the widest layer that there is.
*/
__global__ void
train_example_kernel(const float * feature_vectors,  // feature vector [ni]
                     int feature_vector_width,
                     const int * labels,
                     const float * example_weights,
                     int num_layers,
                     const float * const * w,  // weights for each layer
                     const float * const * biases, // for each layer
                     const int * architecture,
                     const int * w_strides,
                     float * const * w_updates, // wt updates for each layer
                     float * const * b_updates, // bias upd for each layer
                     int activation,            // activation function
                     float fire,   // target value for firing neuron
                     float inhibit, // target value for inhibited neuron)
                     float learning_rate)
{
    // access thread id
    const unsigned tid = threadIdx.x;

    // 
    const unsigned example_num  = blockIdx.x;

    /* The layer outputs (activation of the neurons).  This is where the
       shared memory goes to.  Note that we store only the activated outputs,
       not the inputs. */
    extern __shared__ float layer_outputs[];

    /* Where we accumulate our errors, layer by layer.  The size is that of
       the largest dimension. */
    extern __shared__ float errors[];

    const float * input = feature_vectors + example_num * feature_vector_width;

    int label = labels[example_num];

    float example_weight = example_weights[example_num];

    
    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    const float * last_layer_outputs = 0;
    float * this_layer_outputs = layer_outputs;
    float * next_layer_outputs;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs = this_layer_outputs,
             this_layer_outputs = next_layer_outputs) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        next_layer_outputs = this_layer_outputs + no;

        // Start off with the bias terms
        if (tid < no) this_layer_outputs[tid] = biases[l][tid];

        /* Add in the layer outputs.  We iterate with all threads */
        if (tid < no) {
            float accum = 0;
            for (unsigned i = 0;  i < ni;  ++i) {
                float inval = (l == 0 ? input[i] : last_layer_outputs[i]);

                // Coalesced access; maybe texture would be better
                float weight = layer_weights[i * w_stride + tid];

                accum += weight * inval;
            }

            this_layer_outputs[tid] = transform(accum, activation);
        }
    }

    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];
    
    /* First error calculation pass */
    if (tid < no) {
        bool correct = (label == tid);
        float wanted = (correct ? fire : inhibit);
        errors[tid] = wanted - last_layer_outputs[tid];
    }

    /* Let everything catch up */
    __syncthreads();

    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 1;
         --l,
             __syncthreads(),
             last_layer_outputs = this_layer_outputs,
             this_layer_outputs = next_layer_outputs) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        float * layer_updates = w_updates[l];
        float * layer_bias_updates  = b_updates[l];
        
        next_layer_outputs = this_layer_outputs - no;
        
        if (tid >= no) continue;
        
        float d = delta(last_layer_outputs[tid], errors[tid], activation);

        /* Calculate the new error terms for the next layer */
        // TODO: atomic... and then find a way to avoid data dependencies...
        if (l > 1)
            for (unsigned i = 0;  i < ni;  ++i)
                errors[i] += d * layer_weights[i * w_stride + tid];


        /* Update the weights. */
        float k = example_weight * learning_rate;
        for (unsigned i = 0;  i < ni;  ++i) {
            // No bank conflicts here as all threads are reading with the same
            // i value
            float k2 = last_layer_outputs[i] * k;
            
            layer_updates[i * w_stride + tid] += k2 * d;
        }
        
        /* Update the bias */
        layer_bias_updates[tid] += k * d;
    }
}

namespace ML {
namespace CUDA {

void train_examples(const float * feature_vectors,
                    int num_feature_vectors,
                    int feature_vector_width,
                    const float * example_weights,
                    const int * labels,
                    int num_layers,
                    const int * architecture,
                    const float * const * weights,
                    const float * const * biases,
                    const int * w_strides,
                    float * const * weight_updates,
                    float * const * bias_updates,
                    Activation activation,
                    float fire,
                    float inhibit,
                    float learning_rate)
{
    DeviceData<float> d_feature_vectors
        (feature_vectors,
         num_feature_vectors * feature_vector_width);

    if (feature_vector_width != architecture[0])
        throw Exception("number of inputs doesn't match");

    DeviceData<float> d_example_weights(example_weights,
                                              num_feature_vectors);

    DeviceData<int> d_labels(labels, num_feature_vectors);

    DeviceData<int> d_architecture(architecture, num_layers + 1);

    vector<DeviceData<float> > d_weights_storage(num_layers);
    const float * d_weights[num_layers];

    for (unsigned l = 0;  l < num_layers;  ++l) {
        int no = architecture[l + 1];
        int w_stride = w_strides[l];
        d_weights_storage[l].init(weights[l], no * w_stride);
        d_weights[l] = d_weights_storage[l];
    }
    
    vector<DeviceData<float> > d_biases_storage(num_layers);
    const float * d_biases[num_layers];

    for (unsigned l = 0;  l < num_layers;  ++l) {
        int no = architecture[l + 1];
        d_biases_storage[l].init(biases[l], no);
        d_biases[l] = d_biases_storage[l];
    }
    
    DeviceData<int> d_w_strides(w_strides, num_layers);

    vector<DeviceData<float> > d_weight_updates_storage(num_layers);
    float * d_weight_updates[num_layers];

    for (unsigned l = 0;  l < num_layers;  ++l) {
        int no = architecture[l + 1];
        int w_stride = w_strides[l];
        d_weight_updates_storage[l].init(weights[l], no * w_stride);
        d_weight_updates[l] = d_weight_updates_storage[l];
    }

    vector<DeviceData<float> > d_bias_updates_storage(num_layers);
    float * d_bias_updates[num_layers];

    for (unsigned l = 0;  l < num_layers;  ++l) {
        int no = architecture[l + 1];
        d_bias_updates_storage[l].init(biases[l], no);
        d_bias_updates[l] = d_bias_updates_storage[l];
    }

    int max_width = 0;
    int total_neurons = 0;
    for (unsigned l = 0;  l <= num_layers;  ++l) {
        max_width = max(max_width, architecture[l]);
        total_neurons += architecture[l];
    }

    // We need our grid size to be exactly the maximum width of the output
    dim3 threads(max_width);

    // Our grid size is one per example
    dim3 grid(num_feature_vectors);

    size_t shared_mem_size = (max_width + total_neurons) * sizeof(float);

    train_example_kernel <<< grid, threads, shared_mem_size >>>
        (d_feature_vectors,
         feature_vector_width,
         d_labels,
         d_example_weights,
         num_layers,
         d_weights,
         d_biases,
         d_architecture,
         d_w_strides,
         d_weight_updates,
         d_bias_updates,
         activation,
         fire,
         inhibit,
         learning_rate);
}

} // namespace CUDA
} // namespace ML
