#include "hip/hip_runtime.h"
/* backprop_cuda.cc                                                -*- C++ -*-
   Jeremy Barnes, 25 May 2009
   Copyright (c) 2009 Jeremy Barnes.  All rights reserved.

   CUDA-based backprop implementation.
*/


#include "arch/exception.h"
#include "compiler/compiler.h"
#include <cstdio>
#include <iostream>
#include <boost/timer.hpp>
#include <boost/utility.hpp>
#include "arch/cuda/device_data.h"
#include "arch/cuda/atomic.h"
#include "math/xdiv.h"
#include "perceptron_defs.h"
#include <vector>
#include "backprop_cuda.h"
#include "fixed_point_accum.h"

using namespace std;


/* TODO:
   - Allow it to run with max_width > 512 (maximum thread block width)
   - tanh function that gives bit-for-bit equivalent results as on the
     host
   - Remove learning rate from the update (apply it when updating the weights)
     and use a constant that conditions the numbers to work well within the
     range of the update
   - Process 4 feature vectors together in the same loop
*/


typedef ML::FixedPointAccum32 UpdateFloat;
//typedef float UpdateFloat;

/** Given an activation function and an input, apply that activation
    function */
__device__ float transform(float input, int activation)
{
    switch (activation) {
    case ML::ACT_TANH: {
        float exp2i = __expf(input + input);
        return __fdividef(exp2i - 1.0f, exp2i + 1.0f);
    }
    case ML::ACT_IDENTITY: return input;
    default:
        return 0.0;
    }
}

/** Given an output and an error, what's the delta (derivative * error)? */
__device__ float delta(float output, float error, int activation)
{
    switch (activation) {
    case ML::ACT_TANH:
        return (1.0f - output * output) * error;
    case ML::ACT_IDENTITY: return output * error; 
    default:
        return 0.0;
    }
}

/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    27 multiprocessors with 8 cores each, and so we could train on 27 different
    feature vectors in parallel.
*/
__device__ void
train_example(const float * input,
              int label,
              float example_weight,
              int num_layers,
              float * scratch,  // shared memory scratch space
              const float * const * w,  // weights for each layer
              const float * const * biases, // for each layer
              const int * architecture,
              const int * w_strides,
              UpdateFloat * const * w_updates, // wt updates for each layer
              UpdateFloat * const * b_updates, // bias upd for each layer
              int activation,            // activation function
              float fire,   // target value for firing neuron
              float inhibit, // target value for inhibited neuron)
              float learning_rate,
              int num_threads_in_block,
              int total_neurons,
              float * layer_outputs)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;

    const unsigned block_num  = blockIdx.x;

#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "starting fprop example %d wt %f; label %d\n",
                block_num, example_weight, label);

        int feature_vector_width = architecture[0];

        for (unsigned i = 0;  i < feature_vector_width;  ++i) {
            fprintf(stderr, "input %d: value %f\n",
                    i, input[i]);
        }
    }
#endif


    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0];
    scratch[tid] = (tid < ni ? input[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();


    float * last_layer_outputs = 0;
    float * this_layer_outputs = layer_outputs;
    float * next_layer_outputs;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs = this_layer_outputs,
             this_layer_outputs = next_layer_outputs) {

        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        next_layer_outputs = this_layer_outputs + no;

#if defined(__DEVICE_EMULATION__) && 1
        if (tid == 0)
            fprintf(stderr, "fprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p next_layer_outputs %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs,
                    next_layer_outputs);
#endif

        /* Add in the layer outputs.  We iterate with all threads */
        
        // Start off with the bias terms
        double accum = (tid < no ? biases[l][tid] : 0.0);

        if (__any(tid < no)) {

            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval = scratch[i];
                //float inval = (l == 0 ? input[i] : last_layer_outputs[i]);
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum += weight * inval;
            }
        }         

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no)
                this_layer_outputs[tid] = scratch[tid]
                    = transform(accum, activation);
        }

#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();
        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed fprop layer %d example %d; label %d\n",
                    l, block_num, label);
            for (unsigned i = 0;  i < no;  ++i) {
                fprintf(stderr, "output %d: value %f\n",
                        i, this_layer_outputs[i]);
            }
        }
#endif
        
    }


    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];

    this_layer_outputs = last_layer_outputs;
    
    /* First error calculation pass */
    bool correct = (label == tid);
    float wanted = (correct ? fire : inhibit);

    float last_output = scratch[tid];

    __syncthreads();

    scratch[tid] = (tid < no ? wanted - last_output : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "completed fprop example %d; label %d\n",
                block_num, label);
        for (unsigned i = 0;  i < no;  ++i) {
            fprintf(stderr, "output %d: value %f error %f correct %d\n",
                    i, this_layer_outputs[i], scratch[i], (label == i));
        }
    }
#endif


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             this_layer_outputs = last_layer_outputs) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        last_layer_outputs = this_layer_outputs - ni;

#if defined(__DEVICE_EMULATION__) && 1
        if (tid == 0)
            fprintf(stderr, "bprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p layer_outputs %p end %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs,
                    layer_outputs, layer_outputs + total_neurons);
#endif

        
        float prev_output = (tid >= no ? 0.0 : this_layer_outputs[tid]);

        float error = scratch[tid];
        
        float d = (tid >= no ? 0.0 : delta(prev_output, error, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch[tid] = d;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d = scratch[o];  // may be the d from another thread
                    float update = d * layer_weights[tid * w_stride + o];
                    total += update;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch[tid] = total;
        }


#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();

        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed error propagation layer %d\n",
                    l);
            for (unsigned i = 0;  i < ni;  ++i) {
                fprintf(stderr, "input %d: error %f\n",
                        i, scratch[i]);
            }
        }
#endif


        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k = example_weight * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev = (l == 0 ? input[i] : last_layer_outputs[i]); 
            float update = prev * k * d;

            atomic_add(layer_updates[i * w_stride + tid], update);
        }
        
        /* Update the bias */
        float update = k * d;

        //layer_bias_updates[tid] += update;
        atomic_add(layer_bias_updates[tid], update);
    }
}

/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    27 multiprocessors with 8 cores each, and so we could train on 27 different
    feature vectors in parallel.
*/
__device__ void
train_4_examples(const float * input1,
                 const float * input2,
                 const float * input3,
                 const float * input4,
                 int4 label,
                 float4 example_weight,
                 int num_layers,
                 float * scratch1,
                 float * scratch2,
                 float * scratch3,
                 float * scratch4,
                 const float * const * w,  // weights for each layer
                 const float * const * biases, // for each layer
                 const int * architecture,
                 const int * w_strides,
                 UpdateFloat * const * w_updates, // wt updates for each layer
                 UpdateFloat * const * b_updates, // bias upd for each layer
                 int activation,            // activation function
                 float fire,   // target value for firing neuron
                 float inhibit, // target value for inhibited neuron)
                 float learning_rate,
                 int num_threads_in_block,
                 int total_neurons,
                 float * layer_outputs1,
                 float * layer_outputs2,
                 float * layer_outputs3,
                 float * layer_outputs4)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;
    
    const unsigned block_num  = blockIdx.x;
    

    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0];
    scratch1[tid] = (tid < ni ? input1[tid] : 0.0);
    scratch2[tid] = (tid < ni ? input2[tid] : 0.0);
    scratch3[tid] = (tid < ni ? input3[tid] : 0.0);
    scratch4[tid] = (tid < ni ? input4[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();


    float * last_layer_outputs1 = 0;
    float * this_layer_outputs1 = layer_outputs1;
    float * next_layer_outputs1;

    float * last_layer_outputs2 = 0;
    float * this_layer_outputs2 = layer_outputs2;
    float * next_layer_outputs2;

    float * last_layer_outputs3 = 0;
    float * this_layer_outputs3 = layer_outputs3;
    float * next_layer_outputs3;

    float * last_layer_outputs4 = 0;
    float * this_layer_outputs4 = layer_outputs4;
    float * next_layer_outputs4;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs1 = this_layer_outputs1,
             this_layer_outputs1 = next_layer_outputs1,
             last_layer_outputs2 = this_layer_outputs2,
             this_layer_outputs2 = next_layer_outputs2,
             last_layer_outputs3 = this_layer_outputs3,
             this_layer_outputs3 = next_layer_outputs3,
             last_layer_outputs4 = this_layer_outputs4,
             this_layer_outputs4 = next_layer_outputs4) {

        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        next_layer_outputs1 = this_layer_outputs1 + no;
        next_layer_outputs2 = this_layer_outputs2 + no;
        next_layer_outputs3 = this_layer_outputs3 + no;
        next_layer_outputs4 = this_layer_outputs4 + no;

        /* Add in the layer outputs.  We iterate with all threads */
        
        // Start off with the bias terms
        double accum1 = (tid < no ? biases[l][tid] : 0.0);
        double accum2 = accum1, accum3 = accum1, accum4 = accum1;

        if (__any(tid < no)) {

            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval1 = scratch1[i];
                float inval2 = scratch2[i];
                float inval3 = scratch3[i];
                float inval4 = scratch4[i];
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum1 += weight * inval1;
                accum2 += weight * inval2;
                accum3 += weight * inval3;
                accum4 += weight * inval4;
            }
        }         

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no) {
                this_layer_outputs1[tid] = scratch1[tid]
                    = transform(accum1, activation);
                this_layer_outputs2[tid] = scratch2[tid]
                    = transform(accum1, activation);
                this_layer_outputs3[tid] = scratch3[tid]
                    = transform(accum1, activation);
                this_layer_outputs4[tid] = scratch4[tid]
                    = transform(accum1, activation);
            }
        }
    }


    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];

    this_layer_outputs1 = last_layer_outputs1;
    this_layer_outputs2 = last_layer_outputs2;
    this_layer_outputs3 = last_layer_outputs3;
    this_layer_outputs4 = last_layer_outputs4;
    
    /* First error calculation pass */
    bool correct1 = (label.x == tid);
    bool correct2 = (label.y == tid);
    bool correct3 = (label.z == tid);
    bool correct4 = (label.w == tid);

    float wanted1 = (correct1 ? fire : inhibit);
    float wanted2 = (correct2 ? fire : inhibit);
    float wanted3 = (correct3 ? fire : inhibit);
    float wanted4 = (correct4 ? fire : inhibit);

    float last_output1 = scratch1[tid];
    float last_output2 = scratch2[tid];
    float last_output3 = scratch3[tid];
    float last_output4 = scratch4[tid];

    __syncthreads();

    scratch1[tid] = (tid < no ? wanted1 - last_output1 : 0.0);
    scratch2[tid] = (tid < no ? wanted2 - last_output2 : 0.0);
    scratch3[tid] = (tid < no ? wanted3 - last_output3 : 0.0);
    scratch4[tid] = (tid < no ? wanted4 - last_output4 : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             this_layer_outputs1 = last_layer_outputs1,
             this_layer_outputs2 = last_layer_outputs2,
             this_layer_outputs3 = last_layer_outputs3,
             this_layer_outputs4 = last_layer_outputs4) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        last_layer_outputs1 = this_layer_outputs1 - ni;
        last_layer_outputs2 = this_layer_outputs2 - ni;
        last_layer_outputs3 = this_layer_outputs3 - ni;
        last_layer_outputs4 = this_layer_outputs4 - ni;
        
        float prev_output1 = (tid >= no ? 0.0 : this_layer_outputs1[tid]);
        float prev_output2 = (tid >= no ? 0.0 : this_layer_outputs2[tid]);
        float prev_output3 = (tid >= no ? 0.0 : this_layer_outputs3[tid]);
        float prev_output4 = (tid >= no ? 0.0 : this_layer_outputs4[tid]);

        float error1 = scratch1[tid];
        float error2 = scratch2[tid];
        float error3 = scratch3[tid];
        float error4 = scratch4[tid];
        
        float d1 = (tid >= no ? 0.0 : delta(prev_output1, error1, activation));
        float d2 = (tid >= no ? 0.0 : delta(prev_output2, error2, activation));
        float d3 = (tid >= no ? 0.0 : delta(prev_output3, error3, activation));
        float d4 = (tid >= no ? 0.0 : delta(prev_output4, error4, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch1[tid] = d1;
            scratch2[tid] = d2;
            scratch3[tid] = d3;
            scratch4[tid] = d4;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total1 = 0.0, total2 = 0.0, total3 = 0.0, total4 = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d1 = scratch1[o];
                    float d2 = scratch2[o];
                    float d3 = scratch3[o];
                    float d4 = scratch4[o];
                    
                    float w = layer_weights[tid * w_stride + o];

                    float update1 = d1 * w;
                    float update2 = d2 * w;
                    float update3 = d3 * w;
                    float update4 = d4 * w;

                    total1 += update1;
                    total2 += update2;
                    total3 += update3;
                    total4 += update4;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch1[tid] = total1;
            scratch2[tid] = total2;
            scratch3[tid] = total3;
            scratch4[tid] = total4;
        }

        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k1 = example_weight.x * learning_rate;
        float k2 = example_weight.y * learning_rate;
        float k3 = example_weight.z * learning_rate;
        float k4 = example_weight.w * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev1 = (l == 0 ? input1[i] : last_layer_outputs1[i]); 
            float prev2 = (l == 0 ? input2[i] : last_layer_outputs2[i]); 
            float prev3 = (l == 0 ? input3[i] : last_layer_outputs3[i]); 
            float prev4 = (l == 0 ? input4[i] : last_layer_outputs4[i]); 

#if 1
            float update1 = prev1 * k1 * d1;
            float update2 = prev2 * k2 * d2;
            float update3 = prev3 * k3 * d3;
            float update4 = prev4 * k4 * d4;

            atomic_add(layer_updates[i * w_stride + tid], update1);
            atomic_add(layer_updates[i * w_stride + tid], update2);
            atomic_add(layer_updates[i * w_stride + tid], update3);
            atomic_add(layer_updates[i * w_stride + tid], update4);
#else
            float update
                = (k1 * d1 * prev1)
                + (k2 * d2 * prev2)
                + (k3 * d3 * prev3)
                + (k4 * d4 * prev4);

            atomic_add(layer_updates[i * w_stride + tid], update);
#endif
        }
#if 1
        float update1 = k1 * d1;
        float update2 = k2 * d2;
        float update3 = k3 * d3;
        float update4 = k4 * d4;

        atomic_add(layer_bias_updates[tid], update1);
        atomic_add(layer_bias_updates[tid], update2);
        atomic_add(layer_bias_updates[tid], update3);
        atomic_add(layer_bias_updates[tid], update4);
#else
        /* Update the bias */
        float update = ((k1 * d1) + (k2 * d2) + (k3 * d3) + (k4 * d4));

        atomic_add(layer_bias_updates[tid], update);
#endif
    }

#if 0
        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev = (l == 0 ? input[i] : last_layer_outputs[i]); 
            float update = prev * k * d;

            atomic_add(layer_updates[i * w_stride + tid], update);
        }
        
        /* Update the bias */
        float update = k * d;

        //layer_bias_updates[tid] += update;
        atomic_add(layer_bias_updates[tid], update);
#endif

}

__global__ void
train_examples_kernel(const float * feature_vectors,  // feature vector [ni]
                      int feature_vector_width,
                      const int * labels,
                      const float * example_weights,
                      int num_layers,
                      const float * const * w,  // weights for each layer
                      const float * const * biases, // for each layer
                      const int * architecture,
                      const int * w_strides,
                      UpdateFloat * const * w_updates, // wt updates for each layer
                      UpdateFloat * const * b_updates, // bias upd for each layer
                      int activation,            // activation function
                      float fire,   // target value for firing neuron
                      float inhibit, // target value for inhibited neuron)
                      float learning_rate,
                      int num_threads_in_block,
                      int total_neurons,
                      float * layer_outputs,  // scratch space[total neurons]
                      int examples_per_block,
                      int total_num_examples,
                      int max_width)
{
    const unsigned block_num  = blockIdx.x;
    
    /* Where we accumulate our errors, layer by layer.  The size is that of
       the largest dimension. */
    extern __shared__ float scratch[];
    
    /* The layer outputs (activation of the neurons).  This is where the
       shared memory goes to.  Note that we store only the activated outputs,
       not the inputs.

       blockDim.x gives us the number of threads, which is also the size of
       the errors array, so that our layer outputs have to start at this
       offset.
    */

    // Get our private scratch memory for this block
    layer_outputs += block_num * total_neurons * 4;
    
    unsigned example_num_base = block_num * examples_per_block;
    unsigned last_example = min(total_num_examples, example_num_base + examples_per_block);

    unsigned example_num = example_num_base;
#if 0
    for (;  example_num < last_example - 3;  example_num += 4) {
        const float * input1 = feature_vectors + example_num * feature_vector_width;
        const float * input2 = input1 + feature_vector_width;
        const float * input3 = input2 + feature_vector_width;
        const float * input4 = input3 + feature_vector_width;

        int4 label;
        label.x = labels[example_num];
        label.y = labels[example_num + 1];
        label.z = labels[example_num + 2];
        label.w = labels[example_num + 3];

        float4 example_weight;
        example_weight.x = example_weights[example_num];
        example_weight.y = example_weights[example_num + 1];
        example_weight.z = example_weights[example_num + 2];
        example_weight.w = example_weights[example_num + 3];

        float * scratch1 = scratch;
        float * scratch2 = scratch1 + max_width;
        float * scratch3 = scratch2 + max_width;
        float * scratch4 = scratch3 + max_width;

        float * layer_outputs1 = layer_outputs;
        float * layer_outputs2 = layer_outputs1 + total_neurons;
        float * layer_outputs3 = layer_outputs2 + total_neurons;
        float * layer_outputs4 = layer_outputs3 + total_neurons;
        
        train_4_examples(input1, input2, input3, input4,
                         label, example_weight,
                         num_layers, scratch1, scratch2, scratch3, scratch4,
                         w, biases, architecture, w_strides,
                         w_updates, b_updates,
                         activation, fire, inhibit, learning_rate,
                         num_threads_in_block, total_neurons,
                         layer_outputs1, layer_outputs2, layer_outputs3,
                         layer_outputs4);

    }
#endif

#if 1
    for (;  example_num < last_example;  example_num += 1) {
        const float * input1 = feature_vectors + example_num * feature_vector_width;
        const float * input2 = input1 + feature_vector_width;
        const float * input3 = input1;
        const float * input4 = input2;

        int4 label;
        label.x = labels[example_num];
        label.y = labels[example_num + 1];
        label.z = labels[example_num];
        label.w = labels[example_num + 1];

        float4 example_weight;
        example_weight.x = 0.0;
        example_weight.y = 0.0;
        example_weight.z = example_weights[example_num];
        example_weight.w = 0.0;//example_weights[example_num + 1];

        float * scratch1 = scratch;
        float * scratch2 = scratch1 + max_width;
        float * scratch3 = scratch2 + max_width;
        float * scratch4 = scratch3 + max_width;
        
        float * layer_outputs1 = layer_outputs;
        float * layer_outputs2 = layer_outputs1 + total_neurons;
        float * layer_outputs3 = layer_outputs2 + total_neurons;
        float * layer_outputs4 = layer_outputs3 + total_neurons;
        
        train_4_examples(input1, input2, input3, input4,
                         label, example_weight,
                         num_layers, scratch1, scratch2, scratch3, scratch4,
                         w, biases, architecture, w_strides,
                         w_updates, b_updates,
                         activation, fire, inhibit, learning_rate,
                         num_threads_in_block, total_neurons,
                         layer_outputs1, layer_outputs2, layer_outputs3,
                         layer_outputs4);

    }
#endif

    for (;  example_num < last_example;  ++example_num) {

        const float * input = feature_vectors + example_num * feature_vector_width;

        int label = labels[example_num];

        float example_weight = example_weights[example_num];

        train_example(input, label, example_weight,
                      num_layers, scratch, w, biases, architecture, w_strides,
                      w_updates, b_updates,
                      activation, fire, inhibit, learning_rate,
                      num_threads_in_block,
                      total_neurons, layer_outputs);
    }
}


namespace ML {
namespace CUDA {

struct Backprop::Plan {
    int num_layers;

    vector<int> architecture;
    DeviceData<int> d_architecture;

    vector<DeviceData<float> > d_weights_storage;
    vector<const float *> weights_vec;
    DeviceData<const float *> d_weights;

    vector<DeviceData<float> > d_biases_storage;
    vector<const float *> biases_vec;
    DeviceData<const float *> d_biases;

    vector<int> w_strides;
    DeviceData<int> d_w_strides;

    Activation activation;
    float fire;
    float inhibit;
    float learning_rate;

    int max_width;
    int total_neurons;

    // We need our grid size to be exactly the maximum width of the output
    dim3 threads;
    
    size_t shared_mem_size;

    Plan(int num_layers,
         const int * architecture,
         const float * const * weights,
         const float * const * biases,
         const int * w_strides,
         Activation activation,
         float fire,
         float inhibit,
         float learning_rate,
         bool on_host)
        : num_layers(num_layers),
          architecture(architecture, architecture + num_layers + 1),
          w_strides(w_strides, w_strides + num_layers),
          activation(activation),
          fire(fire),
          inhibit(inhibit),
          learning_rate(learning_rate)
    {
        //cerr << "plan: num_layers = " << num_layers << endl;

        d_architecture.init(architecture, num_layers + 1);

        d_weights_storage.resize(num_layers);
        weights_vec.resize(num_layers);

        for (unsigned l = 0;  l < num_layers;  ++l) {
            int ni = architecture[l];
            int w_stride = w_strides[l];
            d_weights_storage[l].init(weights[l], ni * w_stride);
            weights_vec[l] = d_weights_storage[l];

            //cerr << "layer " << l << ": no = " << no << " w_stride = "
            //     << w_stride << endl;
        }
    
        d_weights.init(&weights_vec[0], num_layers);

        d_biases_storage.resize(num_layers);
        biases_vec.resize(num_layers);

        for (unsigned l = 0;  l < num_layers;  ++l) {
            int no = architecture[l + 1];
            d_biases_storage[l].init(biases[l], no);
            biases_vec[l] = d_biases_storage[l];
        }
    
        d_biases.init(&biases_vec[0], num_layers);

        d_w_strides.init(w_strides, num_layers);
        
        max_width = 0;
        total_neurons = 0;

        for (unsigned l = 0;  l <= num_layers;  ++l) {
            max_width = max(max_width, architecture[l]);
            total_neurons += architecture[l];
        }

        // We need our grid size to be exactly the maximum width of the output
        threads = dim3(max_width);

        // Since we do 4 examples per loop, we need enough memory for all of
        // the four outputs for a single layer
        shared_mem_size = max_width * sizeof(float)* 4;
    }
};

struct Backprop::Context {

    const Plan & plan;
    
    DeviceData<float> d_feature_vectors;
    DeviceData<float> d_example_weights;
    DeviceData<int> d_labels;
        
    float * const * weight_updates;
    float * const * bias_updates;

    vector<DeviceData<UpdateFloat> > d_weight_updates_storage;
    vector<UpdateFloat *> weight_updates_vec;
    DeviceData<UpdateFloat *> d_weight_updates;
    
    vector<DeviceData<UpdateFloat> > d_bias_updates_storage;
    vector<UpdateFloat *> bias_updates_vec;
    DeviceData<UpdateFloat *> d_bias_updates;

    DeviceData<float> d_layer_outputs;

    dim3 grid;

    int num_feature_vectors;
    int feature_vector_width;
    int num_examples_per_invocation;

    Context(const Plan & plan,
            const float * feature_vectors,
            int num_feature_vectors,
            const float * example_weights,
            const int * labels,
            float * const * weight_updates,
            float * const * bias_updates,
            float & correct,
            float & total,
            float & rms_error)
        : plan(plan), weight_updates(weight_updates),
          bias_updates(bias_updates), num_feature_vectors(num_feature_vectors),
          feature_vector_width(feature_vector_width)
    {
        feature_vector_width = plan.architecture[0];
        
        //cerr << "num_feature_vectors = " << num_feature_vectors << endl;
        //cerr << "feature_vector_width = " << feature_vector_width
        //     << endl;

        d_feature_vectors.init(feature_vectors,
                               num_feature_vectors * feature_vector_width);
        
        d_example_weights.init(example_weights, num_feature_vectors);
        
        d_labels.init(labels, num_feature_vectors);
        
        d_weight_updates_storage.resize(plan.num_layers);
        weight_updates_vec.resize(plan.num_layers);
        
        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            d_weight_updates_storage[l].init_zeroed(ni * w_stride);
            weight_updates_vec[l] = d_weight_updates_storage[l];
        }

        d_weight_updates.init(&weight_updates_vec[0], plan.num_layers);

        d_bias_updates_storage.resize(plan.num_layers);
        bias_updates_vec.resize(plan.num_layers);

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];
            d_bias_updates_storage[l].init_zeroed(no);
            bias_updates_vec[l] = d_bias_updates_storage[l];
        }

        d_bias_updates.init(&bias_updates_vec[0], plan.num_layers);

        num_examples_per_invocation = 4;//16;

        int grid_size = rudiv(num_feature_vectors, num_examples_per_invocation);

        // Get the scratch space
        d_layer_outputs.init(plan.total_neurons * grid_size * 4);
        
        // Our grid size is one per example
        grid = dim3(grid_size);
    }

    void execute()
    {
        train_examples_kernel<<<grid, plan.threads, plan.shared_mem_size>>>
            (d_feature_vectors,
             feature_vector_width,
             d_labels,
             d_example_weights,
             plan.num_layers,
             plan.d_weights,
             plan.d_biases,
             plan.d_architecture,
             plan.d_w_strides,
             d_weight_updates,
             d_bias_updates,
             plan.activation,
             plan.fire,
             plan.inhibit,
             plan.learning_rate,
             grid.x,
             plan.total_neurons,
             d_layer_outputs,
             num_examples_per_invocation,
             num_feature_vectors /* total num examples */,
             plan.max_width);

        //cerr << "launched" << endl;
    }
    
    void synchronize()
    {
        //cerr << "waiting for execution" << endl;
        hipError_t err = hipDeviceSynchronize();
        
        if (err != hipSuccess)
            throw Exception(hipGetErrorString(err));

        //cerr << "copying memory back" << endl;

        


        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            
            UpdateFloat sync_to[ni * w_stride];

            d_weight_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + ni * w_stride, weight_updates[l]);
        }

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];

            UpdateFloat sync_to[no];
            
            d_bias_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + no, bias_updates[l]);
        }
    }
};

boost::shared_ptr<Backprop::Plan>
Backprop::
plan(int num_layers,
     const int * architecture,
     const float * const * weights,
     const float * const * biases,
     const int * w_strides,
     Activation activation,
     float fire,
     float inhibit,
     float learning_rate,
     bool on_host) const
{
    boost::shared_ptr<Plan> result
        (new Plan(num_layers, architecture, weights, biases, w_strides,
                  activation, fire, inhibit, learning_rate, on_host));

    return result;
}

boost::shared_ptr<Backprop::Context>
Backprop::
execute(const Plan & plan,
        const float * feature_vectors,
        int num_feature_vectors,
        const float * example_weights,
        const int * labels,
        float * const * weight_updates,
        float * const * bias_updates,
        float & correct,
        float & total,
        float & rms_error) const
{
    boost::shared_ptr<Context> result
        (new Context(plan, feature_vectors, num_feature_vectors,
                     example_weights, labels,
                     weight_updates, bias_updates,
                     correct, total, rms_error));

    result->execute();

    return result;
}

/** Wait for the given context to be finished. */
void
Backprop::
synchronize(Context & context) const
{
    context.synchronize();
}


} // namespace CUDA
} // namespace ML
