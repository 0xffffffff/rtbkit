#include "hip/hip_runtime.h"
/* backprop_cuda.cc                                                -*- C++ -*-
   Jeremy Barnes, 25 May 2009
   Copyright (c) 2009 Jeremy Barnes.  All rights reserved.

   CUDA-based backprop implementation.
*/


#include "arch/exception.h"
#include "compiler/compiler.h"
#include <cstdio>
#include <iostream>
#include <boost/timer.hpp>
#include <boost/utility.hpp>
#include "arch/cuda/device_data.h"
#include "arch/cuda/atomic.h"
#include "math/xdiv.h"
#include "perceptron_defs.h"
#include <vector>
#include "backprop_cuda.h"
#include "fixed_point_accum.h"

using namespace std;


/* TODO:
   - Allow it to run with max_width > 512 (maximum thread block width)
   - tanh function that gives bit-for-bit equivalent results as on the
     host
   - Remove learning rate from the update (apply it when updating the weights)
     and use a constant that conditions the numbers to work well within the
     range of the update
   - Try using textures for the W arrays (caching could make a big difference)
*/


typedef ML::FixedPointAccum32 UpdateFloat;
//typedef float UpdateFloat;

/** Given an activation function and an input, apply that activation
    function */
__device__ float transform(float input, int activation)
{
    switch (activation) {
    case ML::ACT_TANH: {
        return tanh(input);
        //float exp2i = __expf(input + input);
        //return __fdividef(exp2i - 1.0f, exp2i + 1.0f);
    }
    case ML::ACT_IDENTITY: return input;
    default:
        return 0.0;
    }
}

/** Given an output and an error, what's the delta (derivative * error)? */
__device__ float delta(float output, float error, int activation)
{
    switch (activation) {
    case ML::ACT_TANH:
        return (1.0f - output * output) * error;
    case ML::ACT_IDENTITY: return output * error; 
    default:
        return 0.0;
    }
}

texture<float, 1, hipReadModeElementType> weights_tex;
texture<float, 1, hipReadModeElementType> biases_tex;;


#if !defined(__DEVICE_EMULATION__)

template<const texture<float, 1, hipReadModeElementType> & Tex>
struct WeightsAccess {
    const float * base;  // if zero, then texture access
    int offset;

    __device__ WeightsAccess(const float * base = 0)
        : base(base), offset(0)
    {
    }

    __device__ void init(const float * base)
    {
        this->base = base;
        offset = 0;
    }

    __device__ void operator += (int val)
    {
        offset += val;
    }

    __device__ void operator -= (int val)
    {
        offset -= val;
    }

    __device__ float operator [] (int ofs)
    {
        if (base) return base[offset + ofs];
        else return tex1Dfetch(Tex, offset + ofs);
    }
};

#endif

/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    27 multiprocessors with 8 cores each, and so we could train on 27 different
    feature vectors in parallel.
*/

#define N 4
#define train_N_examples train_4_examples
#include "backprop_cuda_train_N_examples.cu"

#define N 1
#define train_N_examples train_1_example
#include "backprop_cuda_train_N_examples.cu"
#undef N
#undef train_N_examples

#if 0
__device__ void
train_example(const float * input,
              int label,
              float example_weight,
              int num_layers,
              float * scratch,  // shared memory scratch space
              const WeightsAccess<weights_tex> & w,
              const WeightsAccess<biases_tex> & biases,
              const int * architecture,
              const int * w_strides,
              UpdateFloat * const * w_updates, // wt updates for each layer
              UpdateFloat * const * b_updates, // bias upd for each layer
              int activation,            // activation function
              float inhibit, // target value for inhibited neuron)
              float fire,   // target value for firing neuron
              float learning_rate,
              int num_threads_in_block,
              int num_threads_on_multiprocessor,
              int total_neurons,
              float * layer_outputs)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;

    const unsigned block_num  = blockIdx.x;

#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "starting fprop example %d wt %f; label %d\n",
                block_num, example_weight, label);

        int feature_vector_width = architecture[0];

        for (unsigned i = 0;  i < feature_vector_width;  ++i) {
            fprintf(stderr, "input %d: value %f\n",
                    i, input[i]);
        }
    }
#endif


    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0], no, w_stride;
    scratch[tid] = (tid < ni ? input[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();

    float * this_layer_outputs = layer_outputs;
    WeightsAccess<weights_tex> layer_weights = w;
    WeightsAccess<biases_tex> layer_biases  = biases;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             layer_weights += ni * w_stride,
             layer_biases += no,
             this_layer_outputs += no) {

        // Get architecture information about the layer:
        ni = architecture[l];
        no = architecture[l + 1];
        w_stride = w_strides[l];

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "fprop: layer %d ni %d no %d this_layer_outputs %p layer_weights %p layer_biases %p\n",
                    l, ni, no, this_layer_outputs,
                    layer_weights, layer_biases);
#endif

        /* Add in the layer outputs.  We iterate with all threads */
        
#if 0
        // Start off with the bias terms
        double accum = (tid < no ? layer_biases[tid] : 0.0);

        if (__any(tid < no)) {

            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval = scratch[i];
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum += weight * inval;
            }
        }         

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no)
                this_layer_outputs[tid]
                    = scratch[tid]
                    = transform(accum, activation);
        }

#else
        /* We want to have each thread working here, even if no is much less
           than the number of threads.  To do so, we assign each thread to
           a certain o value and a certain subset of the i values, and then
           accumulate the updates, broadcasting them at the end.

           For example:
           32 threads
           2 outputs

           So we have 16 threads working on each example

           100 threads
           16 outputs

           So we have 7 threads on the first 4 examples, and 6 threads on
           the rest.
        */

        int nt = num_threads_on_multiprocessor;

        int min_threads = nt / no;
        int left_over   = nt % no;
        int max_threads = min_threads + (left_over > 0);

        int o = tid % no;    // which o value are we working on?
        int idx = tid / no;  // which thread in that block?
        int o_threads = min_threads + (o < left_over);

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "fprop: layer %d ni %d no %d min_threads %d left_over %d max_threads %d nib %d nt %d\n",
                    l, ni, no, min_threads, left_over, max_threads,
                    num_threads_in_block);
        fprintf(stderr, "      tid %d o %d idx %d o_threads %d\n",
                tid, o, idx, o_threads);
#endif

        double accum = 0.0;

        for (unsigned i = idx;  i < ni;  i += o_threads) {
            // warning: bank conflicts...
            float inval = scratch[i];
            float weight = layer_weights[i * w_stride + o];
            
            accum += weight * inval;
        }

        if (max_threads > 1) {

            __syncthreads();

            if (tid < no) scratch[tid] = layer_biases[tid];

            __syncthreads();
            
            /* Now we accumulate them, allowing each thread to increment in its
               turn. */
            for (unsigned i = 0;  i < max_threads;  ++i, __syncthreads())
                if (i == idx) scratch[o] += accum;
            
            if (__any(tid < no)) {
                
                if (tid < no)
                    this_layer_outputs[tid]
                        = scratch[tid]
                        = transform(scratch[tid], activation);
            }
        }
        else {
            accum += layer_biases[o];
            this_layer_outputs[o]
                = scratch[o]
                = transform(accum, activation);
            
        }
#endif


#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();
        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed fprop layer %d example %d; label %d\n",
                    l, block_num, label);
            for (unsigned i = 0;  i < no;  ++i) {
                fprintf(stderr, "output %d: value %f\n",
                        i, this_layer_outputs[i]);
            }
        }
#endif
        
    }

    // layer_biases is no longer used

    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    this_layer_outputs -= no;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "before fixing up layer_weights: %p no: %d w_stride: %d\n",
                    layer_weights, no, w_stride);
#endif

    layer_weights -= ni * w_stride;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "after fixing up layer_weights: %p no: %d w_stride: %d\n",
                    layer_weights, no, w_stride);
#endif

    /* First error calculation pass */
    bool correct = (label == tid);
    float wanted = (correct ? fire : inhibit);

    float last_output = scratch[tid];

    __syncthreads();

    scratch[tid] = (tid < no ? wanted - last_output : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "completed fprop example %d; label %d\n",
                block_num, label);
        for (unsigned i = 0;  i < no;  ++i) {
            fprintf(stderr, "output %d: value %f error %f correct %d\n",
                    i, this_layer_outputs[i], scratch[i], (label == i));
        }
    }
#endif


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             layer_weights -= (l == -1 ? 0 : architecture[l] * w_strides[l]),
             this_layer_outputs -= architecture[l + 1]) {
        
        // Get information about the layer:
        ni = architecture[l];
        no = architecture[l + 1];
        w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        const float * last_layer_outputs = this_layer_outputs - ni;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "bprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p layer_outputs %p end %p layer_weights %p w_stride %d new layer_weights %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs,
                    layer_outputs, layer_outputs + total_neurons,
                    layer_weights, w_stride, layer_weights - (ni * w_stride));
#endif

        
        float prev_output = (tid >= no ? 0.0 : this_layer_outputs[tid]);

        float error = scratch[tid];
        
        float d = (tid >= no ? 0.0 : delta(prev_output, error, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch[tid] = d;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d = scratch[o];  // may be the d from another thread
                    float update = d * layer_weights[tid * w_stride + o];
                    total += update;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch[tid] = total;
        }


#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();

        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed error propagation layer %d\n",
                    l);
            for (unsigned i = 0;  i < ni;  ++i) {
                fprintf(stderr, "input %d: error %f\n",
                        i, scratch[i]);
            }
        }
#endif


        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k = example_weight * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev = (l == 0 ? input[i] : last_layer_outputs[i]); 
            float update = prev * k * d;

#if defined(__DEVICE_EMULATION__) && 0
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev, k, d, update, (float)layer_updates[i * w_stride + tid]);
            }
#endif

            atomic_add(layer_updates[i * w_stride + tid], update);

#if defined(__DEVICE_EMULATION__) && 0
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "                          after %.15g\n", (float)layer_updates[i * w_stride + tid]);
            }
#endif
        }
        
        /* Update the bias */
        float update = k * d;

        //layer_bias_updates[tid] += update;
        atomic_add(layer_bias_updates[tid], update);
    }
}
#endif


__global__ void
train_examples_kernel(const float * feature_vectors,  // feature vector [ni]
                      int feature_vector_width,
                      const int * labels,
                      const float * example_weights,
                      int num_layers,
                      const float * w,  // weights for each layer
                      const float * biases, // for each layer
                      const int * architecture,
                      const int * w_strides,
                      UpdateFloat * const * w_updates, // wt updates for each layer
                      UpdateFloat * const * b_updates, // bias upd for each layer
                      int activation,            // activation function
                      float inhibit, // target value for inhibited neuron)
                      float fire,   // target value for firing neuron
                      float learning_rate,
                      int num_threads_in_block,
                      int num_threads_on_multiprocessor,
                      int total_neurons,
                      float * layer_outputs,  // scratch space[total neurons]
                      int examples_per_block,
                      int total_num_examples,
                      int max_width,
                      bool use_textures)
{
    const unsigned block_num  = blockIdx.x;
    
    /* Where we accumulate our errors, layer by layer.  The size is that of
       the largest dimension. */
    extern __shared__ float scratch[];
    
    /* The layer outputs (activation of the neurons).  This is where the
       shared memory goes to.  Note that we store only the activated outputs,
       not the inputs.

       blockDim.x gives us the number of threads, which is also the size of
       the errors array, so that our layer outputs have to start at this
       offset.
    */

    // Get our private scratch memory for this block
    layer_outputs += block_num * total_neurons;
    
    unsigned example_num_base = block_num * examples_per_block;
    unsigned last_example = min(total_num_examples, example_num_base + examples_per_block);

    unsigned example_num = example_num_base;

#if defined(__DEVICE_EMULATION__)
    const float * weights_access = w;
    const float * biases_access = biases;
#else
    WeightsAccess<weights_tex> weights_access;
    WeightsAccess<biases_tex> biases_access;

    if (!use_textures) {
        weights_access.init(w);
        biases_access.init(biases);
    }
#endif

#if 0
    for (;  example_num < last_example;  example_num += N) {

        const float * input = feature_vectors + example_num * feature_vector_width;
        train_4_examples(input,
                         labels + example_num,
                         example_weights + example_num,
                         last_example - example_num,
                         num_layers, scratch,
                         weights_access, biases_access,
                         architecture, w_strides,
                         w_updates, b_updates,
                         activation, inhibit, fire, learning_rate,
                         num_threads_in_block,
                         num_threads_on_multiprocessor,
                         total_neurons, max_width, layer_outputs);
    }
#elif 0
    // Do any others singly
    for (;  example_num < last_example;  ++example_num) {

        const float * input
            = feature_vectors + example_num * feature_vector_width;

        train_4_examples(input,
                         labels + example_num,
                         example_weights + example_num,
                         1 /* num valid examples */,
                         num_layers, scratch,
                         weights_access, biases_access,
                         architecture, w_strides,
                         w_updates, b_updates,
                         activation, inhibit, fire, learning_rate,
                         num_threads_in_block,
                         num_threads_on_multiprocessor,
                         total_neurons, max_width, layer_outputs);
    }
#else
    // Do any others singly
    for (;  example_num < last_example;  ++example_num) {

        const float * input
            = feature_vectors + example_num * feature_vector_width;

        train_1_example(input,
                        labels + example_num,
                        example_weights + example_num,
                        1 /* num valid examples */,
                        num_layers, scratch,
                        weights_access, biases_access,
                        architecture, w_strides,
                        w_updates, b_updates,
                        activation, inhibit, fire, learning_rate,
                        num_threads_in_block,
                        num_threads_on_multiprocessor,
                        total_neurons, max_width, layer_outputs);
    }
#endif
}


namespace ML {
namespace CUDA {

struct Backprop::Plan {
    int num_layers;

    vector<int> architecture;
    DeviceData<int> d_architecture;

    DeviceData<float> d_weights;

    DeviceData<float> d_biases;

    vector<int> w_strides;
    DeviceData<int> d_w_strides;

    Activation activation;
    float inhibit;
    float fire;
    float learning_rate;

    int max_width;
    int total_neurons;

    // We need our grid size to be exactly the maximum width of the output
    dim3 threads;
    
    int shared_mem_stride;
    size_t shared_mem_size;

    bool use_textures;

    Plan(int num_layers,
         const int * architecture,
         const float * const * weights,
         const float * const * biases,
         const int * w_strides,
         Activation activation,
         float inhibit,
         float fire,
         float learning_rate,
         bool on_host,
         bool use_textures)
        : num_layers(num_layers),
          architecture(architecture, architecture + num_layers + 1),
          w_strides(w_strides, w_strides + num_layers),
          activation(activation),
          inhibit(inhibit),
          fire(fire),
          learning_rate(learning_rate),
          use_textures(use_textures)
    {
        //cerr << "plan: num_layers = " << num_layers << endl;

        d_architecture.init(architecture, num_layers + 1);

        size_t total_weights_size = 0;
        size_t total_bias_size = 0;

        for (unsigned l = 0;  l < num_layers;  ++l) {
            int ni = architecture[l];
            int no = architecture[l + 1];
            int w_stride = w_strides[l];
            total_weights_size += ni * w_stride;
            total_bias_size += no;
            // TODO: align?
        }

        d_weights.init(total_weights_size);
        d_biases.init(total_bias_size);
        
        // Now copy them all in

        size_t weights_start_offset = 0;
        size_t bias_start_offset = 0;
        
        for (unsigned l = 0;  l < num_layers;  ++l) {
            int ni = architecture[l];
            int no = architecture[l + 1];
            int w_stride = w_strides[l];
            size_t w_size = ni * w_stride;

            hipError_t err
                = hipMemcpy(d_weights + weights_start_offset,
                             weights[l],
                             w_size * sizeof(float),
                             hipMemcpyHostToDevice);
            
            if (err != hipSuccess)
                throw Exception(hipGetErrorString(err));

            err = hipMemcpy(d_biases + bias_start_offset,
                             biases[l],
                             no * sizeof(float),
                             hipMemcpyHostToDevice);
            
            if (err != hipSuccess)
                throw Exception(hipGetErrorString(err));
            
            weights_start_offset += ni * w_stride;
            bias_start_offset += no;
            // TODO: align?
        }

        d_w_strides.init(w_strides, num_layers);
        
        max_width = 0;
        total_neurons = 0;

        for (unsigned l = 0;  l <= num_layers;  ++l) {
            max_width = max(max_width, architecture[l]);
            total_neurons += architecture[l];
        }

        // We need our grid size to be exactly the maximum width of the output
        threads = dim3(max_width);

        // Storage for max_width
        shared_mem_stride = max_width * sizeof(float);
        
        // Since we do 4 examples per loop, we need enough memory for all of
        // the four outputs for a single layer
        shared_mem_size = shared_mem_stride * 4;

        if (use_textures) {
            hipError_t err;
            
            err = hipBindTexture(0, weights_tex, d_weights);
            if (err != hipSuccess)
                throw Exception(hipGetErrorString(err));

            err = hipBindTexture(0, biases_tex, d_biases);
            if (err != hipSuccess)
                throw Exception(hipGetErrorString(err));
        }
    }
};

struct Backprop::Context {

    const Plan & plan;
    
    DeviceData<float> d_feature_vectors;
    DeviceData<float> d_example_weights;
    DeviceData<int> d_labels;
        
    float * const * weight_updates;
    float * const * bias_updates;

    vector<DeviceData<UpdateFloat> > d_weight_updates_storage;
    vector<UpdateFloat *> weight_updates_vec;
    DeviceData<UpdateFloat *> d_weight_updates;
    
    vector<DeviceData<UpdateFloat> > d_bias_updates_storage;
    vector<UpdateFloat *> bias_updates_vec;
    DeviceData<UpdateFloat *> d_bias_updates;

    DeviceData<float> d_layer_outputs;

    dim3 grid;

    int num_feature_vectors;
    int feature_vector_width;
    int num_examples_per_invocation;

    Context(const Plan & plan,
            const float * feature_vectors,
            int num_feature_vectors,
            const float * example_weights,
            const int * labels,
            float * const * weight_updates,
            float * const * bias_updates,
            float & correct,
            float & total,
            float & rms_error)
        : plan(plan), weight_updates(weight_updates),
          bias_updates(bias_updates), num_feature_vectors(num_feature_vectors),
          feature_vector_width(feature_vector_width)
    {
        feature_vector_width = plan.architecture[0];
        
        //cerr << "num_feature_vectors = " << num_feature_vectors << endl;
        //cerr << "feature_vector_width = " << feature_vector_width
        //     << endl;

        d_feature_vectors.init(feature_vectors,
                               num_feature_vectors * feature_vector_width);
        
        d_example_weights.init(example_weights, num_feature_vectors);
        
        d_labels.init(labels, num_feature_vectors);
        
        d_weight_updates_storage.resize(plan.num_layers);
        weight_updates_vec.resize(plan.num_layers);
        
        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            d_weight_updates_storage[l].init_zeroed(ni * w_stride);
            weight_updates_vec[l] = d_weight_updates_storage[l];
        }

        d_weight_updates.init(&weight_updates_vec[0], plan.num_layers);

        d_bias_updates_storage.resize(plan.num_layers);
        bias_updates_vec.resize(plan.num_layers);

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];
            d_bias_updates_storage[l].init_zeroed(no);
            bias_updates_vec[l] = d_bias_updates_storage[l];
        }

        d_bias_updates.init(&bias_updates_vec[0], plan.num_layers);

        num_examples_per_invocation = 4;//16;

        int grid_size = rudiv(num_feature_vectors, num_examples_per_invocation);

        // Get the scratch space.  This is 4 in flight examples for each
        // of the concurrent threads.
        d_layer_outputs.init(plan.total_neurons * grid_size * 4);
        
        // Our grid size is one per example
        grid = dim3(grid_size);
    }

    void execute()
    {
        train_examples_kernel<<<grid, plan.threads, plan.shared_mem_size>>>
            (d_feature_vectors,
             feature_vector_width,
             d_labels,
             d_example_weights,
             plan.num_layers,
             plan.d_weights,
             plan.d_biases,
             plan.d_architecture,
             plan.d_w_strides,
             d_weight_updates,
             d_bias_updates,
             plan.activation,
             plan.inhibit,
             plan.fire,
             plan.learning_rate,
             grid.x,
             plan.threads.x,
             plan.total_neurons,
             d_layer_outputs,
             num_examples_per_invocation,
             num_feature_vectors /* total num examples */,
             plan.max_width,
             plan.use_textures);
        //cerr << "launched" << endl;
    }
    
    void synchronize()
    {
        //cerr << "waiting for execution" << endl;
        hipError_t err = hipDeviceSynchronize();
        
        if (err != hipSuccess)
            throw Exception(hipGetErrorString(err));

        //cerr << "copying memory back" << endl;

        


        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            
            UpdateFloat sync_to[ni * w_stride];

            d_weight_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + ni * w_stride, weight_updates[l]);

#if 0
            cerr << "first 10 weight updates for layer " << l << ": ";
            for (unsigned i = 0;  i < 10;  ++i)
                cerr << sync_to[i] << " ";
            cerr << endl;
#endif
        }

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];

            UpdateFloat sync_to[no];
            
            d_bias_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + no, bias_updates[l]);

#if 0            
            cerr << "first 10 bias updates for layer " << l << ": ";
            for (unsigned i = 0;  i < 10;  ++i)
                cerr << sync_to[i] << " ";
            cerr << endl;
#endif
        }
    }
};

boost::shared_ptr<Backprop::Plan>
Backprop::
plan(int num_layers,
     const int * architecture,
     const float * const * weights,
     const float * const * biases,
     const int * w_strides,
     Activation activation,
     float inhibit,
     float fire,
     float learning_rate,
     bool on_host,
     bool use_textures) const
{
    boost::shared_ptr<Plan> result
        (new Plan(num_layers, architecture, weights, biases, w_strides,
                  activation, inhibit, fire, learning_rate, on_host,
                  use_textures));

    return result;
}

boost::shared_ptr<Backprop::Context>
Backprop::
execute(const Plan & plan,
        const float * feature_vectors,
        int num_feature_vectors,
        const float * example_weights,
        const int * labels,
        float * const * weight_updates,
        float * const * bias_updates,
        float & correct,
        float & total,
        float & rms_error) const
{
    boost::shared_ptr<Context> result
        (new Context(plan, feature_vectors, num_feature_vectors,
                     example_weights, labels,
                     weight_updates, bias_updates,
                     correct, total, rms_error));

    result->execute();

    return result;
}

/** Wait for the given context to be finished. */
void
Backprop::
synchronize(Context & context) const
{
    context.synchronize();
}


} // namespace CUDA
} // namespace ML
