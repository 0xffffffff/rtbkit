#include "hip/hip_runtime.h"
/* backprop_cuda.cc                                                -*- C++ -*-
   Jeremy Barnes, 25 May 2009
   Copyright (c) 2009 Jeremy Barnes.  All rights reserved.

   CUDA-based backprop implementation.
*/


#include "arch/exception.h"
#include "compiler/compiler.h"
#include <cstdio>
#include <iostream>
#include <boost/timer.hpp>
#include <boost/utility.hpp>
#include "arch/cuda/device_data.h"
#include "arch/cuda/atomic.h"
#include "math/xdiv.h"
#include "perceptron_defs.h"
#include <vector>
#include "backprop_cuda.h"
#include "fixed_point_accum.h"

using namespace std;


typedef ML::FixedPointAccum32 UpdateFloat;
//typedef float UpdateFloat;

/** Given an activation function and an input, apply that activation
    function */
__device__ float transform(float input, int activation)
{
    switch (activation) {
    case ML::ACT_TANH: {
        float exp2i = __expf(input + input);
        return __fdividef(exp2i - 1.0f, exp2i + 1.0f);
    }
    case ML::ACT_IDENTITY: return input;
    default:
        return 0.0;
    }
}

/** Given an output and an error, what's the delta (derivative * error)? */
__device__ float delta(float output, float error, int activation)
{
    switch (activation) {
    case ML::ACT_TANH:
        return (1.0f - output * output) * error;
    case ML::ACT_IDENTITY: return output * error; 
    default:
        return 0.0;
    }
}

/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    28 multiprocessors with 8 cores each, and so we could compute 28 different
    samples at once).
*/
__global__ void
train_example_kernel(const float * feature_vectors,  // feature vector [ni]
                     int feature_vector_width,
                     const int * labels,
                     const float * example_weights,
                     int num_layers,
                     const float * const * w,  // weights for each layer
                     const float * const * biases, // for each layer
                     const int * architecture,
                     const int * w_strides,
                     UpdateFloat * const * w_updates, // wt updates for each layer
                     UpdateFloat * const * b_updates, // bias upd for each layer
                     int activation,            // activation function
                     float fire,   // target value for firing neuron
                     float inhibit, // target value for inhibited neuron)
                     float learning_rate,
                     int num_threads_in_block)
{
    // access thread id
    const unsigned tid = threadIdx.x;

    const unsigned example_num  = blockIdx.x;

#ifdef __DEVICE_EMULATION__
    //fprintf(stderr, "tid = %d example_num = %d\n",
    //        tid, example_num);
#endif

    /* Where we accumulate our errors, layer by layer.  The size is that of
       the largest dimension. */
    extern __shared__ float errors[];

    /* The layer outputs (activation of the neurons).  This is where the
       shared memory goes to.  Note that we store only the activated outputs,
       not the inputs.

       blockDim.x gives us the number of threads, which is also the size of
       the errors array, so that our layer outputs have to start at this
       offset.
    */
    float * layer_outputs = errors + blockDim.x;

    const float * input = feature_vectors + example_num * feature_vector_width;

    int label = labels[example_num];

    float example_weight = example_weights[example_num];

#ifdef __DEVICE_EMULATION__
    if (tid == 0 && example_num == 0) {
        fprintf(stderr, "starting fprop example %d wt %f; label %d\n",
                example_num, example_weight, label);

        for (unsigned i = 0;  i < feature_vector_width;  ++i) {
            fprintf(stderr, "input %d: value %f\n",
                    i, input[i]);
        }
    }
#endif


    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    float * last_layer_outputs = 0;
    float * this_layer_outputs = layer_outputs;
    float * next_layer_outputs;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs = this_layer_outputs,
             this_layer_outputs = next_layer_outputs) {

        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        next_layer_outputs = this_layer_outputs + no;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "fprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs);
#endif

#if 0
    std::copy(bias.begin(), bias.end(), output.begin());
    for (unsigned i = 0;  i < input.size();  ++i)
        SIMD::vec_add(&output[0], input[i], &weights[i][0], &output[0],
                      outputs());
        //for (unsigned o = 0;  o < output.size();  ++o)
        //    output[o] += input[i] * weights[i][o];
    transform(output);
#endif

        /* Add in the layer outputs.  We iterate with all threads */
        if (tid < no) {
            // Start off with the bias terms
            double accum = biases[l][tid];

            for (unsigned i = 0;  i < ni;  ++i) {
                float inval = (l == 0 ? input[i] : last_layer_outputs[i]);

                // Coalesced access; maybe texture would be better
                float weight = layer_weights[i * w_stride + tid];
                
                accum += weight * inval;
            }
            
            this_layer_outputs[tid] = transform(accum, activation);
        }

#if defined(__DEVICE_EMULATION__)
        __syncthreads();
        if (tid == 0 && example_num == 0) {
            fprintf(stderr, "completed fprop layer %d example %d; label %d\n",
                    l, example_num, label);
            for (unsigned i = 0;  i < no;  ++i) {
                fprintf(stderr, "output %d: value %f\n",
                        i, this_layer_outputs[i]);
            }
        }
#endif
        
    }


    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];

    this_layer_outputs = last_layer_outputs;
    
    /* First error calculation pass */
    bool correct = (label == tid);
    float wanted = (correct ? fire : inhibit);
    errors[tid] = (tid < no ? wanted - this_layer_outputs[tid] : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


#if defined(__DEVICE_EMULATION__)
    if (tid == 0 && example_num == 0) {
        fprintf(stderr, "completed fprop example %d; label %d\n",
                example_num, label);
        for (unsigned i = 0;  i < no;  ++i) {
            fprintf(stderr, "output %d: value %f error %f correct %d\n",
                    i, this_layer_outputs[i], errors[i], (label == i));
        }
    }
#endif


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             this_layer_outputs = last_layer_outputs) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        last_layer_outputs = this_layer_outputs - ni;
        
        float prev_output = (tid > no ? 0.0 : last_layer_outputs[tid]);

        if (prev_output > 1.0) prev_output *= 1.0000001;

        // 
        float error = errors[tid];
        
        float d = (tid > no ? 0.0 : delta(prev_output, error, activation));
        float d2 = 0.0;

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            errors[tid] = d;

            // Make sure everything can get its d value
            __syncthreads();

            // Get the d value for the next thread, for the updates
            if (tid < no - 1) d2 = errors[tid + 1];
            
            double total = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d = errors[o];  // may be the d from another thread
                    float update = d * layer_weights[tid * w_stride + o];
                    total += update;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            errors[tid] = total;
        }


#if defined(__DEVICE_EMULATION__)
        __syncthreads();

        if (tid == 0 && example_num == 0) {
            fprintf(stderr, "completed error propagation layer %d\n",
                    l);
            for (unsigned i = 0;  i < ni;  ++i) {
                fprintf(stderr, "input %d: error %f\n",
                        i, errors[i]);
            }
        }
#endif


        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k = example_weight * learning_rate;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "bprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs);
#endif

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (example_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {
            if (tid % 2 == 1) continue;

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;
            
            float prev = (l == 0 ? input[i] : last_layer_outputs[i]); 
            float update = prev * k * d;
            float update2 = prev * k * d2;

            ML::FixedPointAccum32 upd1(update), upd2(update2);

            unsigned long long two_updates
                = (((unsigned long long)upd2.rep) << 32)
                | upd1.rep;
            
            atomicAdd((unsigned long long *)(layer_updates
                                             + i * w_stride + tid),
                      two_updates);
            
            //atomic_add(layer_updates[i * w_stride + tid], update);
        }
        
        /* Update the bias */
        float update = k * d;

        //layer_bias_updates[tid] += update;
        atomic_add(layer_bias_updates[tid], update);
    }
}

namespace ML {
namespace CUDA {

struct Backprop::Plan {
    int num_layers;

    vector<int> architecture;
    DeviceData<int> d_architecture;

    vector<DeviceData<float> > d_weights_storage;
    vector<const float *> weights_vec;
    DeviceData<const float *> d_weights;

    vector<DeviceData<float> > d_biases_storage;
    vector<const float *> biases_vec;
    DeviceData<const float *> d_biases;

    vector<int> w_strides;
    DeviceData<int> d_w_strides;

    Activation activation;
    float fire;
    float inhibit;
    float learning_rate;

    int max_width;
    int total_neurons;

    // We need our grid size to be exactly the maximum width of the output
    dim3 threads;
    
    size_t shared_mem_size;

    Plan(int num_layers,
         const int * architecture,
         const float * const * weights,
         const float * const * biases,
         const int * w_strides,
         Activation activation,
         float fire,
         float inhibit,
         float learning_rate,
         bool on_host)
        : num_layers(num_layers),
          architecture(architecture, architecture + num_layers + 1),
          w_strides(w_strides, w_strides + num_layers),
          activation(activation),
          fire(fire),
          inhibit(inhibit),
          learning_rate(learning_rate)
    {
        //cerr << "plan: num_layers = " << num_layers << endl;

        d_architecture.init(architecture, num_layers + 1);

        d_weights_storage.resize(num_layers);
        weights_vec.resize(num_layers);

        for (unsigned l = 0;  l < num_layers;  ++l) {
            int ni = architecture[l];
            int w_stride = w_strides[l];
            d_weights_storage[l].init(weights[l], ni * w_stride);
            weights_vec[l] = d_weights_storage[l];

            //cerr << "layer " << l << ": no = " << no << " w_stride = "
            //     << w_stride << endl;
        }
    
        d_weights.init(&weights_vec[0], num_layers);

        d_biases_storage.resize(num_layers);
        biases_vec.resize(num_layers);

        for (unsigned l = 0;  l < num_layers;  ++l) {
            int no = architecture[l + 1];
            d_biases_storage[l].init(biases[l], no);
            biases_vec[l] = d_biases_storage[l];
        }
    
        d_biases.init(&biases_vec[0], num_layers);

        d_w_strides.init(w_strides, num_layers);
        
        max_width = 0;
        total_neurons = 0;

        for (unsigned l = 0;  l <= num_layers;  ++l) {
            max_width = max(max_width, architecture[l]);
            total_neurons += architecture[l];
        }

        // We need our grid size to be exactly the maximum width of the output
        threads = dim3(max_width);

        shared_mem_size = (max_width + total_neurons) * sizeof(float);
    }
};

struct Backprop::Context {

    const Plan & plan;
    
    DeviceData<float> d_feature_vectors;
    DeviceData<float> d_example_weights;
    DeviceData<int> d_labels;
        
    float * const * weight_updates;
    float * const * bias_updates;

    vector<DeviceData<UpdateFloat> > d_weight_updates_storage;
    vector<UpdateFloat *> weight_updates_vec;
    DeviceData<UpdateFloat *> d_weight_updates;
    
    vector<DeviceData<UpdateFloat> > d_bias_updates_storage;
    vector<UpdateFloat *> bias_updates_vec;
    DeviceData<UpdateFloat *> d_bias_updates;

    dim3 grid;

    int num_feature_vectors;
    int feature_vector_width;

    Context(const Plan & plan,
            const float * feature_vectors,
            int num_feature_vectors,
            const float * example_weights,
            const int * labels,
            float * const * weight_updates,
            float * const * bias_updates,
            float & correct,
            float & total,
            float & rms_error)
        : plan(plan), weight_updates(weight_updates),
          bias_updates(bias_updates), num_feature_vectors(num_feature_vectors),
          feature_vector_width(feature_vector_width)
    {
        feature_vector_width = plan.architecture[0];
        
        //cerr << "num_feature_vectors = " << num_feature_vectors << endl;
        //cerr << "feature_vector_width = " << feature_vector_width
        //     << endl;

        d_feature_vectors.init(feature_vectors,
                               num_feature_vectors * feature_vector_width);
        
        d_example_weights.init(example_weights, num_feature_vectors);
        
        d_labels.init(labels, num_feature_vectors);
        
        d_weight_updates_storage.resize(plan.num_layers);
        weight_updates_vec.resize(plan.num_layers);
        
        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            d_weight_updates_storage[l].init(ni * w_stride);
            weight_updates_vec[l] = d_weight_updates_storage[l];
        }

        d_weight_updates.init(&weight_updates_vec[0], plan.num_layers);

        d_bias_updates_storage.resize(plan.num_layers);
        bias_updates_vec.resize(plan.num_layers);

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];
            d_bias_updates_storage[l].init(no);
            bias_updates_vec[l] = d_bias_updates_storage[l];
        }

        d_bias_updates.init(&bias_updates_vec[0], plan.num_layers);

        int total_inputs = 0;
        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            total_inputs += ni;
        }
        
        // Our grid size is one per example
        grid = dim3(num_feature_vectors);
    }

    void execute()
    {
        train_example_kernel<<<grid, plan.threads, plan.shared_mem_size>>>
            (d_feature_vectors,
             feature_vector_width,
             d_labels,
             d_example_weights,
             plan.num_layers,
             plan.d_weights,
             plan.d_biases,
             plan.d_architecture,
             plan.d_w_strides,
             d_weight_updates,
             d_bias_updates,
             plan.activation,
             plan.fire,
             plan.inhibit,
             plan.learning_rate,
             num_feature_vectors);

        //cerr << "launched" << endl;
    }
    
    void synchronize()
    {
        //cerr << "waiting for execution" << endl;
        hipError_t err = hipDeviceSynchronize();
        
        if (err != hipSuccess)
            throw Exception(hipGetErrorString(err));

        //cerr << "copying memory back" << endl;

        


        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            
            UpdateFloat sync_to[ni * w_stride];

            d_weight_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + ni * w_stride, weight_updates[l]);
        }

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];

            UpdateFloat sync_to[no];
            
            d_bias_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + no, bias_updates[l]);
        }
    }
};

boost::shared_ptr<Backprop::Plan>
Backprop::
plan(int num_layers,
     const int * architecture,
     const float * const * weights,
     const float * const * biases,
     const int * w_strides,
     Activation activation,
     float fire,
     float inhibit,
     float learning_rate,
     bool on_host) const
{
    boost::shared_ptr<Plan> result
        (new Plan(num_layers, architecture, weights, biases, w_strides,
                  activation, fire, inhibit, learning_rate, on_host));

    return result;
}

boost::shared_ptr<Backprop::Context>
Backprop::
execute(const Plan & plan,
        const float * feature_vectors,
        int num_feature_vectors,
        const float * example_weights,
        const int * labels,
        float * const * weight_updates,
        float * const * bias_updates,
        float & correct,
        float & total,
        float & rms_error) const
{
    boost::shared_ptr<Context> result
        (new Context(plan, feature_vectors, num_feature_vectors,
                     example_weights, labels,
                     weight_updates, bias_updates,
                     correct, total, rms_error));

    result->execute();

    return result;
}

/** Wait for the given context to be finished. */
void
Backprop::
synchronize(Context & context) const
{
    context.synchronize();
}


} // namespace CUDA
} // namespace ML
