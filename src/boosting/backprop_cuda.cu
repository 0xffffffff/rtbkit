#include "hip/hip_runtime.h"
/* backprop_cuda.cc                                                -*- C++ -*-
   Jeremy Barnes, 25 May 2009
   Copyright (c) 2009 Jeremy Barnes.  All rights reserved.

   CUDA-based backprop implementation.
*/


#include "arch/exception.h"
#include "compiler/compiler.h"
#include <cstdio>
#include <iostream>
#include <boost/timer.hpp>
#include <boost/utility.hpp>
#include "arch/cuda/device_data.h"
#include "arch/cuda/atomic.h"
#include "math/xdiv.h"
#include "perceptron_defs.h"
#include <vector>
#include "backprop_cuda.h"
#include "fixed_point_accum.h"

using namespace std;


/* TODO:
   - Allow it to run with max_width > 512 (maximum thread block width)
   - tanh function that gives bit-for-bit equivalent results as on the
     host
   - Remove learning rate from the update (apply it when updating the weights)
     and use a constant that conditions the numbers to work well within the
     range of the update
   - Try using textures for the W arrays (caching could make a big difference)
*/


typedef ML::FixedPointAccum32 UpdateFloat;
//typedef float UpdateFloat;

/** Given an activation function and an input, apply that activation
    function */
__device__ float transform(float input, int activation)
{
    switch (activation) {
    case ML::ACT_TANH: {
        return tanh(input);
        //float exp2i = __expf(input + input);
        //return __fdividef(exp2i - 1.0f, exp2i + 1.0f);
    }
    case ML::ACT_IDENTITY: return input;
    default:
        return 0.0;
    }
}

/** Given an output and an error, what's the delta (derivative * error)? */
__device__ float delta(float output, float error, int activation)
{
    switch (activation) {
    case ML::ACT_TANH:
        return (1.0f - output * output) * error;
    case ML::ACT_IDENTITY: return output * error; 
    default:
        return 0.0;
    }
}


/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    27 multiprocessors with 8 cores each, and so we could train on 27 different
    feature vectors in parallel.
*/
__device__ void
train_example(const float * input,
              int label,
              float example_weight,
              int num_layers,
              float * scratch,  // shared memory scratch space
              const float * w,  // weights for each layer, packed
              const float * biases, // for each layer
              const int * architecture,
              const int * w_strides,
              UpdateFloat * const * w_updates, // wt updates for each layer
              UpdateFloat * const * b_updates, // bias upd for each layer
              int activation,            // activation function
              float fire,   // target value for firing neuron
              float inhibit, // target value for inhibited neuron)
              float learning_rate,
              int num_threads_in_block,
              int num_threads_on_multiprocessor,
              int total_neurons,
              float * layer_outputs)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;

    const unsigned block_num  = blockIdx.x;

#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "starting fprop example %d wt %f; label %d\n",
                block_num, example_weight, label);

        int feature_vector_width = architecture[0];

        for (unsigned i = 0;  i < feature_vector_width;  ++i) {
            fprintf(stderr, "input %d: value %f\n",
                    i, input[i]);
        }
    }
#endif


    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0], no, w_stride;
    scratch[tid] = (tid < ni ? input[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();

    float * this_layer_outputs = layer_outputs;
    const float * layer_weights = w;
    const float * layer_biases  = biases;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             layer_weights += ni * w_stride,
             layer_biases += no,
             this_layer_outputs += no) {

        // Get architecture information about the layer:
        ni = architecture[l];
        no = architecture[l + 1];
        w_stride = w_strides[l];

#if defined(__DEVICE_EMULATION__) && 1
        if (tid == 0)
            fprintf(stderr, "fprop: layer %d ni %d no %d this_layer_outputs %p layer_weights %p layer_biases %p\n",
                    l, ni, no, this_layer_outputs,
                    layer_weights, layer_biases);
#endif

        /* Add in the layer outputs.  We iterate with all threads */
        
#if 0
        // Start off with the bias terms
        double accum = (tid < no ? layer_biases[tid] : 0.0);

        if (__any(tid < no)) {

            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval = scratch[i];
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum += weight * inval;
            }
        }         

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no)
                this_layer_outputs[tid]
                    = scratch[tid]
                    = transform(accum, activation);
        }

#else
        /* We want to have each thread working here, even if no is much less
           than the number of threads.  To do so, we assign each thread to
           a certain o value and a certain subset of the i values, and then
           accumulate the updates, broadcasting them at the end.

           For example:
           32 threads
           2 outputs

           So we have 16 threads working on each example

           100 threads
           16 outputs

           So we have 7 threads on the first 4 examples, and 6 threads on
           the rest.
        */

        int nt = num_threads_on_multiprocessor;

        int min_threads = nt / no;
        int left_over   = nt % no;
        int max_threads = min_threads + (left_over > 0);

        int o = tid % no;    // which o value are we working on?
        int idx = tid / no;  // which thread in that block?
        int o_threads = min_threads + (o < left_over);

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "fprop: layer %d ni %d no %d min_threads %d left_over %d max_threads %d nib %d nt %d\n",
                    l, ni, no, min_threads, left_over, max_threads,
                    num_threads_in_block);
        fprintf(stderr, "      tid %d o %d idx %d o_threads %d\n",
                tid, o, idx, o_threads);
#endif

        double accum = 0.0;

        for (unsigned i = idx;  i < ni;  i += o_threads) {
            // warning: bank conflicts...
            float inval = scratch[i];
            float weight = layer_weights[i * w_stride + o];
            
            accum += weight * inval;
        }

        if (max_threads > 1) {

            __syncthreads();

            if (tid < no) scratch[tid] = layer_biases[tid];

            __syncthreads();
            
            /* Now we accumulate them, allowing each thread to increment in its
               turn. */
            for (unsigned i = 0;  i < max_threads;  ++i, __syncthreads())
                if (i == idx) scratch[o] += accum;
            
            __syncthreads();
            
            if (__any(tid < no)) {
                
                if (tid < no)
                    this_layer_outputs[tid]
                        = scratch[tid]
                        = transform(scratch[tid], activation);
            }
        }
        else {
            accum += layer_biases[o];
            this_layer_outputs[o]
                = scratch[o]
                = transform(accum, activation);
            
        }
#endif


#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();
        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed fprop layer %d example %d; label %d\n",
                    l, block_num, label);
            for (unsigned i = 0;  i < no;  ++i) {
                fprintf(stderr, "output %d: value %f\n",
                        i, this_layer_outputs[i]);
            }
        }
#endif
        
    }

    // layer_biases is no longer used

    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    this_layer_outputs -= no;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "before fixing up layer_weights: %p no: %d w_stride: %d\n",
                    layer_weights, no, w_stride);
#endif

    layer_weights -= ni * w_stride;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "after fixing up layer_weights: %p no: %d w_stride: %d\n",
                    layer_weights, no, w_stride);
#endif

    /* First error calculation pass */
    bool correct = (label == tid);
    float wanted = (correct ? fire : inhibit);

    float last_output = scratch[tid];

    __syncthreads();

    scratch[tid] = (tid < no ? wanted - last_output : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "completed fprop example %d; label %d\n",
                block_num, label);
        for (unsigned i = 0;  i < no;  ++i) {
            fprintf(stderr, "output %d: value %f error %f correct %d\n",
                    i, this_layer_outputs[i], scratch[i], (label == i));
        }
    }
#endif


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             layer_weights -= (l == -1 ? 0 : architecture[l] * w_strides[l]),
             this_layer_outputs -= architecture[l + 1]) {
        
        // Get information about the layer:
        ni = architecture[l];
        no = architecture[l + 1];
        w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        const float * last_layer_outputs = this_layer_outputs - ni;

#if defined(__DEVICE_EMULATION__) && 1
        if (tid == 0)
            fprintf(stderr, "bprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p layer_outputs %p end %p layer_weights %p w_stride %d new layer_weights %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs,
                    layer_outputs, layer_outputs + total_neurons,
                    layer_weights, w_stride, layer_weights - (ni * w_stride));
#endif

        
        float prev_output = (tid >= no ? 0.0 : this_layer_outputs[tid]);

        float error = scratch[tid];
        
        float d = (tid >= no ? 0.0 : delta(prev_output, error, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch[tid] = d;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d = scratch[o];  // may be the d from another thread
                    float update = d * layer_weights[tid * w_stride + o];
                    total += update;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch[tid] = total;
        }


#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();

        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed error propagation layer %d\n",
                    l);
            for (unsigned i = 0;  i < ni;  ++i) {
                fprintf(stderr, "input %d: error %f\n",
                        i, scratch[i]);
            }
        }
#endif


        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k = example_weight * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev = (l == 0 ? input[i] : last_layer_outputs[i]); 
            float update = prev * k * d;

#if defined(__DEVICE_EMULATION__) && 0
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev, k, d, update, (float)layer_updates[i * w_stride + tid]);
            }
#endif

            atomic_add(layer_updates[i * w_stride + tid], update);

#if defined(__DEVICE_EMULATION__) && 0
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "                          after %.15g\n", (float)layer_updates[i * w_stride + tid]);
            }
#endif
        }
        
        /* Update the bias */
        float update = k * d;

        //layer_bias_updates[tid] += update;
        atomic_add(layer_bias_updates[tid], update);
    }
}

#if 0
texture<float, 1, hipReadModeElementType> weights_tex;
texture<float, 1, hipReadModeElementType> biases_tex;;

/** Train a fully-connected neural network architecture via backpropagation
    one a single training example.  The work is split over all of the cores
    within a single multiprocessor.  (So, on a Geforce 260 core 216, we have
    27 multiprocessors with 8 cores each, and so we could train on 27 different
    feature vectors in parallel.
*/
__device__ void
train_example_tex(const float * input,
                  int label,
                  float example_weight,
                  int num_layers,
                  float * scratch,  // shared memory scratch space
                  const int * architecture,
                  const int * w_strides,
                  UpdateFloat * const * w_updates, // wt updates for each layer
                  UpdateFloat * const * b_updates, // bias upd for each layer
                  int activation,            // activation function
                  float fire,   // target value for firing neuron
                  float inhibit, // target value for inhibited neuron)
                  float learning_rate,
                  int num_threads_in_block,
                  int num_threads_on_multiprocessor,
                  int total_neurons,
                  float * layer_outputs)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;

    const unsigned block_num  = blockIdx.x;

#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "starting fprop example %d wt %f; label %d\n",
                block_num, example_weight, label);

        int feature_vector_width = architecture[0];

        for (unsigned i = 0;  i < feature_vector_width;  ++i) {
            fprintf(stderr, "input %d: value %f\n",
                    i, input[i]);
        }
    }
#endif


    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0];
    scratch[tid] = (tid < ni ? input[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();


    float * last_layer_outputs = 0;
    float * this_layer_outputs = layer_outputs;
    float * next_layer_outputs;

    int layer_weights_offset = 0;
    int next_layer_weights_offset = 0;
    int layer_bias_offset = 0;
    int next_layer_bias_offset = 0;
    
    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs = this_layer_outputs,
             this_layer_outputs = next_layer_outputs,
             layer_weights_offset = next_layer_weights_offset,
             layer_bias_offset = next_layer_bias_offset) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        int w_stride = w_strides[l];

        next_layer_outputs = this_layer_outputs + no;
        next_layer_weights_offset += ni * w_stride;
        next_layer_bias_offset += no;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "fprop: layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p next_layer_outputs %p\n",
                    l, ni, no, last_layer_outputs, this_layer_outputs,
                    next_layer_outputs);
#endif

        /* Add in the layer outputs.  We iterate with all threads */
        
#if 0
        // Start off with the bias terms
        double accum = (tid < no ? biases[l][tid] : 0.0);

        if (__any(tid < no)) {
            
            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval = scratch[i];
                //float inval = (l == 0 ? input[i] : last_layer_outputs[i]);
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum += weight * inval;
            }
        }         

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no)
                this_layer_outputs[tid]
                    = scratch[tid]
                    = transform(accum, activation);
        }

#else
        /* We want to have each thread working here, even if no is much less
           than the number of threads.  To do so, we assign each thread to
           a certain o value and a certain subset of the i values, and then
           accumulate the updates, broadcasting them at the end.

           For example:
           32 threads
           2 outputs

           So we have 16 threads working on each example

           100 threads
           16 outputs

           So we have 7 threads on the first 4 examples, and 6 threads on
           the rest.
        */

        int nt = num_threads_on_multiprocessor;

        int min_threads = nt / no;
        int left_over   = nt % no;
        int max_threads = min_threads + (left_over > 0);

        int o = tid % no;    // which o value are we working on?
        int idx = tid / no;  // which thread in that block?
        int o_threads = min_threads + (o < left_over);

#if defined(__DEVICE_EMULATION__)
        if (tid == 0)
            fprintf(stderr, "fprop: layer %d ni %d no %d min_threads %d left_over %d max_threads %d nib %d nt %d\n",
                    l, ni, no, min_threads, left_over, max_threads,
                    num_threads_in_block);
        fprintf(stderr, "      tid %d o %d idx %d o_threads %d\n",
                tid, o, idx, o_threads);
#endif

        double accum = 0.0;

        for (unsigned i = idx;  i < ni;  i += o_threads) {
            // warning: bank conflicts...
            float inval = scratch[i];
            float weight = tex1Dfetch(weights_tex,
                                      layer_weights_offset
                                      + i * w_stride + 0);
            
            accum += weight * inval;
        }

        if (max_threads > 1) {

            __syncthreads();

            if (tid < no)
                scratch[tid] = tex1Dfetch(biases_tex,
                                          layer_bias_offset
                                          + tid);
            
            __syncthreads();
            
            /* Now we accumulate them, allowing each thread to increment in its
               turn. */
            for (unsigned i = 0;  i < max_threads;  ++i, __syncthreads())
                if (i == idx) scratch[o] += accum;
            
            __syncthreads();
            
            if (__any(tid < no)) {
                
                if (tid < no)
                    this_layer_outputs[tid]
                        = scratch[tid]
                        = transform(scratch[tid], activation);
            }
        }
        else {
            accum += tex1Dfetch(biases_tex,
                                layer_bias_offset
                                + o);
            this_layer_outputs[o]
                = scratch[o]
                = transform(accum, activation);
            
        }
#endif


#if defined(__DEVICE_EMULATION__) && 1
        __syncthreads();
        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed fprop layer %d example %d; label %d\n",
                    l, block_num, label);
            for (unsigned i = 0;  i < no;  ++i) {
                fprintf(stderr, "output %d: value %f\n",
                        i, this_layer_outputs[i]);
            }
        }
#endif
        
    }


    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];

    this_layer_outputs = last_layer_outputs;
    
    /* First error calculation pass */
    bool correct = (label == tid);
    float wanted = (correct ? fire : inhibit);

    float last_output = scratch[tid];

    __syncthreads();

    scratch[tid] = (tid < no ? wanted - last_output : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


#if defined(__DEVICE_EMULATION__) && 0
    if (tid == 0 && block_num == 0) {
        fprintf(stderr, "completed fprop example %d; label %d\n",
                block_num, label);
        for (unsigned i = 0;  i < no;  ++i) {
            fprintf(stderr, "output %d: value %f error %f correct %d\n",
                    i, this_layer_outputs[i], scratch[i], (label == i));
        }
    }
#endif


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             this_layer_outputs = last_layer_outputs) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        int w_stride = w_strides[l];

        next_weight_offset -= w_stride * ni;
        next_bias_offset -= no;

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        last_layer_outputs = this_layer_outputs - ni;

#if defined(__DEVICE_EMULATION__) && 0
        if (tid == 0)
            fprintf(stderr, "bprop: tid %d layer %d ni %d no %d last_layer_outputs %p this_layer_outputs %p layer_outputs %p end %p\n",
                    tid, l, ni, no, last_layer_outputs, this_layer_outputs,
                    layer_outputs, layer_outputs + total_neurons);
#endif

        
        float prev_output = (tid >= no ? 0.0 : this_layer_outputs[tid]);

        float error = scratch[tid];
        
        float d = (tid >= no ? 0.0 : delta(prev_output, error, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch[tid] = d;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d = scratch[o];  // may be the d from another thread
                    float w = tex1Dfetch(weights_tex,
                                         layer_weights_offset
                                         + tid * w_stride + o);
                    float update = d * w;
                    total += update;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch[tid] = total;
        }


#if defined(__DEVICE_EMULATION__) && 0
        __syncthreads();

        if (tid == 0 && block_num == 0) {
            fprintf(stderr, "completed error propagation layer %d\n",
                    l);
            for (unsigned i = 0;  i < ni;  ++i) {
                fprintf(stderr, "input %d: error %f\n",
                        i, scratch[i]);
            }
        }
#endif


        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k = example_weight * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev = (l == 0 ? input[i] : last_layer_outputs[i]); 
            float update = prev * k * d;

#if defined(__DEVICE_EMULATION__)
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev, k, d, update, (float)layer_updates[i * w_stride + tid]);
            }
#endif

            atomic_add(layer_updates[i * w_stride + tid], update);

#if defined(__DEVICE_EMULATION__)
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "                          after %.15g\n", (float)layer_updates[i * w_stride + tid]);
            }
#endif
        }
        
        /* Update the bias */
        float update = k * d;

        //layer_bias_updates[tid] += update;
        atomic_add(layer_bias_updates[tid], update);
    }
}
#endif

#if 0
/** Train 4 examples at once */
__device__ void
train_4_examples(const float * input1,
                 const float * input2,
                 const float * input3,
                 const float * input4,
                 int4 label,
                 float4 example_weight,
                 int num_layers,
                 float * scratch1,
                 float * scratch2,
                 float * scratch3,
                 float * scratch4,
                 const float * const * w,  // weights for each layer
                 const float * const * biases, // for each layer
                 const int * architecture,
                 const int * w_strides,
                 UpdateFloat * const * w_updates, // wt updates for each layer
                 UpdateFloat * const * b_updates, // bias upd for each layer
                 int activation,            // activation function
                 float fire,   // target value for firing neuron
                 float inhibit, // target value for inhibited neuron)
                 float learning_rate,
                 int num_threads_in_block,
                 int total_neurons,
                 float * layer_outputs1,
                 float * layer_outputs2,
                 float * layer_outputs3,
                 float * layer_outputs4)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;
    
    const unsigned block_num  = blockIdx.x;
    

    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0];
    scratch1[tid] = (tid < ni ? input1[tid] : 0.0);
    scratch2[tid] = (tid < ni ? input2[tid] : 0.0);
    scratch3[tid] = (tid < ni ? input3[tid] : 0.0);
    scratch4[tid] = (tid < ni ? input4[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();


    float * last_layer_outputs1 = 0;
    float * this_layer_outputs1 = layer_outputs1;
    float * next_layer_outputs1;

    float * last_layer_outputs2 = 0;
    float * this_layer_outputs2 = layer_outputs2;
    float * next_layer_outputs2;

    float * last_layer_outputs3 = 0;
    float * this_layer_outputs3 = layer_outputs3;
    float * next_layer_outputs3;

    float * last_layer_outputs4 = 0;
    float * this_layer_outputs4 = layer_outputs4;
    float * next_layer_outputs4;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs1 = this_layer_outputs1,
             this_layer_outputs1 = next_layer_outputs1,
             last_layer_outputs2 = this_layer_outputs2,
             this_layer_outputs2 = next_layer_outputs2,
             last_layer_outputs3 = this_layer_outputs3,
             this_layer_outputs3 = next_layer_outputs3,
             last_layer_outputs4 = this_layer_outputs4,
             this_layer_outputs4 = next_layer_outputs4) {

        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        next_layer_outputs1 = this_layer_outputs1 + no;
        next_layer_outputs2 = this_layer_outputs2 + no;
        next_layer_outputs3 = this_layer_outputs3 + no;
        next_layer_outputs4 = this_layer_outputs4 + no;

        /* Add in the layer outputs.  We iterate with all threads */
        
        // Start off with the bias terms
        double accum1 = (tid < no ? biases[l][tid] : 0.0);
        double accum2 = accum1, accum3 = accum1, accum4 = accum1;

        if (__any(tid < no)) {

            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval1 = scratch1[i];
                float inval2 = scratch2[i];
                float inval3 = scratch3[i];
                float inval4 = scratch4[i];
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum1 += weight * inval1;
                accum2 += weight * inval2;
                accum3 += weight * inval3;
                accum4 += weight * inval4;
            }
        }         

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no) {
                this_layer_outputs1[tid] = scratch1[tid]
                    = transform(accum1, activation);
                this_layer_outputs2[tid] = scratch2[tid]
                    = transform(accum2, activation);
                this_layer_outputs3[tid] = scratch3[tid]
                    = transform(accum3, activation);
                this_layer_outputs4[tid] = scratch4[tid]
                    = transform(accum4, activation);
            }
        }
    }


    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];

    this_layer_outputs1 = last_layer_outputs1;
    this_layer_outputs2 = last_layer_outputs2;
    this_layer_outputs3 = last_layer_outputs3;
    this_layer_outputs4 = last_layer_outputs4;
    
    /* First error calculation pass */
    bool correct1 = (label.x == tid);
    bool correct2 = (label.y == tid);
    bool correct3 = (label.z == tid);
    bool correct4 = (label.w == tid);

    float wanted1 = (correct1 ? fire : inhibit);
    float wanted2 = (correct2 ? fire : inhibit);
    float wanted3 = (correct3 ? fire : inhibit);
    float wanted4 = (correct4 ? fire : inhibit);

    float last_output1 = scratch1[tid];
    float last_output2 = scratch2[tid];
    float last_output3 = scratch3[tid];
    float last_output4 = scratch4[tid];

    __syncthreads();

    scratch1[tid] = (tid < no ? wanted1 - last_output1 : 0.0);
    scratch2[tid] = (tid < no ? wanted2 - last_output2 : 0.0);
    scratch3[tid] = (tid < no ? wanted3 - last_output3 : 0.0);
    scratch4[tid] = (tid < no ? wanted4 - last_output4 : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             this_layer_outputs1 = last_layer_outputs1,
             this_layer_outputs2 = last_layer_outputs2,
             this_layer_outputs3 = last_layer_outputs3,
             this_layer_outputs4 = last_layer_outputs4) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        last_layer_outputs1 = this_layer_outputs1 - ni;
        last_layer_outputs2 = this_layer_outputs2 - ni;
        last_layer_outputs3 = this_layer_outputs3 - ni;
        last_layer_outputs4 = this_layer_outputs4 - ni;
        
        float prev_output1 = (tid >= no ? 0.0 : this_layer_outputs1[tid]);
        float prev_output2 = (tid >= no ? 0.0 : this_layer_outputs2[tid]);
        float prev_output3 = (tid >= no ? 0.0 : this_layer_outputs3[tid]);
        float prev_output4 = (tid >= no ? 0.0 : this_layer_outputs4[tid]);

        float error1 = scratch1[tid];
        float error2 = scratch2[tid];
        float error3 = scratch3[tid];
        float error4 = scratch4[tid];
        
        float d1 = (tid >= no ? 0.0 : delta(prev_output1, error1, activation));
        float d2 = (tid >= no ? 0.0 : delta(prev_output2, error2, activation));
        float d3 = (tid >= no ? 0.0 : delta(prev_output3, error3, activation));
        float d4 = (tid >= no ? 0.0 : delta(prev_output4, error4, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch1[tid] = d1;
            scratch2[tid] = d2;
            scratch3[tid] = d3;
            scratch4[tid] = d4;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total1 = 0.0, total2 = 0.0, total3 = 0.0, total4 = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d1 = scratch1[o];
                    float d2 = scratch2[o];
                    float d3 = scratch3[o];
                    float d4 = scratch4[o];
                    
                    float w = layer_weights[tid * w_stride + o];

                    float update1 = d1 * w;
                    float update2 = d2 * w;
                    float update3 = d3 * w;
                    float update4 = d4 * w;

                    total1 += update1;
                    total2 += update2;
                    total3 += update3;
                    total4 += update4;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch1[tid] = total1;
            scratch2[tid] = total2;
            scratch3[tid] = total3;
            scratch4[tid] = total4;
        }

        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k1 = example_weight.x * learning_rate;
        float k2 = example_weight.y * learning_rate;
        float k3 = example_weight.z * learning_rate;
        float k4 = example_weight.w * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev1 = (l == 0 ? input1[i] : last_layer_outputs1[i]); 
            float prev2 = (l == 0 ? input2[i] : last_layer_outputs2[i]); 
            float prev3 = (l == 0 ? input3[i] : last_layer_outputs3[i]); 
            float prev4 = (l == 0 ? input4[i] : last_layer_outputs4[i]); 

            float update1 = k1 * d1 * prev1;
            float update2 = k2 * d2 * prev2;
            float update3 = k3 * d3 * prev3;
            float update4 = k4 * d4 * prev4;

            float update = update1 + update2 + update3 + update4;

#if defined(__DEVICE_EMULATION__)
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev1, k1, d1, update1, (float)layer_updates[i * w_stride + tid]);
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev2, k2, d2, update2, (float)layer_updates[i * w_stride + tid]);
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev3, k3, d3, update3, (float)layer_updates[i * w_stride + tid]);
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev4, k4, d4, update4, (float)layer_updates[i * w_stride + tid]);
            }
#endif

            atomic_add(layer_updates[i * w_stride + tid], update);

#if defined(__DEVICE_EMULATION__)
            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "                          after %.15g\n", (float)layer_updates[i * w_stride + tid]);
            }
#endif

        }

        /* Update the bias */
        double update
            = double(k1 * d1)
            + double(k2 * d2)
            + double(k3 * d3)
            + double(k4 * d4);

        atomic_add(layer_bias_updates[tid], update);
    }
}
#endif

#if 0
/** Train 8 examples at once */
__device__ void
train_8_examples(const float * input1,
                 const float * input2,
                 const float * input3,
                 const float * input4,
                 const float * input5,
                 const float * input6,
                 const float * input7,
                 const float * input8,
                 int4 label1, int4 label2,
                 float4 example_weight1, float4 example_weight2,
                 int num_layers,
                 float * scratch1,
                 float * scratch2,
                 float * scratch3,
                 float * scratch4,
                 float * scratch5,
                 float * scratch6,
                 float * scratch7,
                 float * scratch8,
                 const float * const * w,  // weights for each layer
                 const float * const * biases, // for each layer
                 const int * architecture,
                 const int * w_strides,
                 UpdateFloat * const * w_updates, // wt updates for each layer
                 UpdateFloat * const * b_updates, // bias upd for each layer
                 int activation,            // activation function
                 float fire,   // target value for firing neuron
                 float inhibit, // target value for inhibited neuron)
                 float learning_rate,
                 int num_threads_in_block,
                 int total_neurons,
                 float * layer_outputs1,
                 float * layer_outputs2,
                 float * layer_outputs3,
                 float * layer_outputs4,
                 float * layer_outputs5,
                 float * layer_outputs6,
                 float * layer_outputs7,
                 float * layer_outputs8)  // global scratch space[total neurons]
{
    // access thread id
    const unsigned tid = threadIdx.x;
    
    const unsigned block_num  = blockIdx.x;
    

    /*************************************************************************/
    /* FPROP                                                                 */
    /*************************************************************************/

    /* First, copy the inputs into shared memory */
    int ni = architecture[0];
    scratch1[tid] = (tid < ni ? input1[tid] : 0.0);
    scratch2[tid] = (tid < ni ? input2[tid] : 0.0);
    scratch3[tid] = (tid < ni ? input3[tid] : 0.0);
    scratch4[tid] = (tid < ni ? input4[tid] : 0.0);
    scratch5[tid] = (tid < ni ? input5[tid] : 0.0);
    scratch6[tid] = (tid < ni ? input6[tid] : 0.0);
    scratch7[tid] = (tid < ni ? input7[tid] : 0.0);
    scratch8[tid] = (tid < ni ? input8[tid] : 0.0);

    /* Let everything catch up */
    __syncthreads();


    float * last_layer_outputs1 = 0;
    float * this_layer_outputs1 = layer_outputs1;
    float * next_layer_outputs1;

    float * last_layer_outputs2 = 0;
    float * this_layer_outputs2 = layer_outputs2;
    float * next_layer_outputs2;

    float * last_layer_outputs3 = 0;
    float * this_layer_outputs3 = layer_outputs3;
    float * next_layer_outputs3;

    float * last_layer_outputs4 = 0;
    float * this_layer_outputs4 = layer_outputs4;
    float * next_layer_outputs4;

    float * last_layer_outputs5 = 0;
    float * this_layer_outputs5 = layer_outputs5;
    float * next_layer_outputs5;

    float * last_layer_outputs6 = 0;
    float * this_layer_outputs6 = layer_outputs6;
    float * next_layer_outputs6;

    float * last_layer_outputs7 = 0;
    float * this_layer_outputs7 = layer_outputs7;
    float * next_layer_outputs7;

    float * last_layer_outputs8 = 0;
    float * this_layer_outputs8 = layer_outputs8;
    float * next_layer_outputs8;

    for (unsigned l = 0;
         l < num_layers;
         ++l,
             __syncthreads(),
             last_layer_outputs1 = this_layer_outputs1,
             this_layer_outputs1 = next_layer_outputs1,
             last_layer_outputs2 = this_layer_outputs2,
             this_layer_outputs2 = next_layer_outputs2,
             last_layer_outputs3 = this_layer_outputs3,
             this_layer_outputs3 = next_layer_outputs3,
             last_layer_outputs4 = this_layer_outputs4,
             this_layer_outputs4 = next_layer_outputs4,
             last_layer_outputs5 = this_layer_outputs5,
             this_layer_outputs5 = next_layer_outputs5,
             last_layer_outputs6 = this_layer_outputs6,
             this_layer_outputs6 = next_layer_outputs6,
             last_layer_outputs7 = this_layer_outputs7,
             this_layer_outputs7 = next_layer_outputs7,
             last_layer_outputs8 = this_layer_outputs8,
             this_layer_outputs8 = next_layer_outputs8
         ) {

        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        next_layer_outputs1 = this_layer_outputs1 + no;
        next_layer_outputs2 = this_layer_outputs2 + no;
        next_layer_outputs3 = this_layer_outputs3 + no;
        next_layer_outputs4 = this_layer_outputs4 + no;
        next_layer_outputs5 = this_layer_outputs5 + no;
        next_layer_outputs6 = this_layer_outputs6 + no;
        next_layer_outputs7 = this_layer_outputs7 + no;
        next_layer_outputs8 = this_layer_outputs8 + no;

        /* Add in the layer outputs.  We iterate with all threads */
        
        // Start off with the bias terms
        double accum1 = (tid < no ? biases[l][tid] : 0.0);
        double accum2 = accum1, accum3 = accum1, accum4 = accum1,
            accum5 = accum1, accum6 = accum1, accum7 = accum1, accum8 = accum1;

        if (__any(tid < no)) {

            for (unsigned i = 0;  i < ni;  ++i) {
                // No bank conflicts as all threads are accessing same value
                float inval1 = scratch1[i];
                float inval2 = scratch2[i];
                float inval3 = scratch3[i];
                float inval4 = scratch4[i];
                float inval5 = scratch5[i];
                float inval6 = scratch6[i];
                float inval7 = scratch7[i];
                float inval8 = scratch8[i];
                
                // Coalesced access; maybe texture would be better
                float weight
                    = (tid < no ? layer_weights[i * w_stride + tid] : 0.0);
                
                accum1 += weight * inval1;
                accum2 += weight * inval2;
                accum3 += weight * inval3;
                accum4 += weight * inval4;
                accum5 += weight * inval5;
                accum6 += weight * inval6;
                accum7 += weight * inval7;
                accum8 += weight * inval8;
            }
        }

        // Let everything catch up so that we can write to scratch
        __syncthreads();
        
        if (__any(tid < no)) {

            if (tid < no) {
                this_layer_outputs1[tid] = scratch1[tid]
                    = transform(accum1, activation);
                this_layer_outputs2[tid] = scratch2[tid]
                    = transform(accum2, activation);
                this_layer_outputs3[tid] = scratch3[tid]
                    = transform(accum3, activation);
                this_layer_outputs4[tid] = scratch4[tid]
                    = transform(accum4, activation);
                this_layer_outputs5[tid] = scratch5[tid]
                    = transform(accum5, activation);
                this_layer_outputs6[tid] = scratch6[tid]
                    = transform(accum6, activation);
                this_layer_outputs7[tid] = scratch7[tid]
                    = transform(accum7, activation);
                this_layer_outputs8[tid] = scratch8[tid]
                    = transform(accum8, activation);
            }
        }
    }


    /*************************************************************************/
    /* BPROP                                                                 */
    /*************************************************************************/

    /* How many output layers? */
    int no = architecture[num_layers];

    this_layer_outputs1 = last_layer_outputs1;
    this_layer_outputs2 = last_layer_outputs2;
    this_layer_outputs3 = last_layer_outputs3;
    this_layer_outputs4 = last_layer_outputs4;
    this_layer_outputs5 = last_layer_outputs5;
    this_layer_outputs6 = last_layer_outputs6;
    this_layer_outputs7 = last_layer_outputs7;
    this_layer_outputs8 = last_layer_outputs8;
    
    /* First error calculation pass */
    bool correct1 = (label1.x == tid);
    bool correct2 = (label1.y == tid);
    bool correct3 = (label1.z == tid);
    bool correct4 = (label1.w == tid);
    bool correct5 = (label2.x == tid);
    bool correct6 = (label2.y == tid);
    bool correct7 = (label2.z == tid);
    bool correct8 = (label2.w == tid);

    float wanted1 = (correct1 ? fire : inhibit);
    float wanted2 = (correct2 ? fire : inhibit);
    float wanted3 = (correct3 ? fire : inhibit);
    float wanted4 = (correct4 ? fire : inhibit);
    float wanted5 = (correct5 ? fire : inhibit);
    float wanted6 = (correct6 ? fire : inhibit);
    float wanted7 = (correct7 ? fire : inhibit);
    float wanted8 = (correct8 ? fire : inhibit);

    float last_output1 = scratch1[tid];
    float last_output2 = scratch2[tid];
    float last_output3 = scratch3[tid];
    float last_output4 = scratch4[tid];
    float last_output5 = scratch5[tid];
    float last_output6 = scratch6[tid];
    float last_output7 = scratch7[tid];
    float last_output8 = scratch8[tid];

    __syncthreads();

    scratch1[tid] = (tid < no ? wanted1 - last_output1 : 0.0);
    scratch2[tid] = (tid < no ? wanted2 - last_output2 : 0.0);
    scratch3[tid] = (tid < no ? wanted3 - last_output3 : 0.0);
    scratch4[tid] = (tid < no ? wanted4 - last_output4 : 0.0);
    scratch5[tid] = (tid < no ? wanted5 - last_output5 : 0.0);
    scratch6[tid] = (tid < no ? wanted6 - last_output6 : 0.0);
    scratch7[tid] = (tid < no ? wanted7 - last_output7 : 0.0);
    scratch8[tid] = (tid < no ? wanted8 - last_output8 : 0.0);
    
    /* Let everything catch up */
    __syncthreads();


    /* Backpropegate. */
    for (int l = num_layers - 1;  l >= 0;
         --l,
             __syncthreads(),
             this_layer_outputs1 = last_layer_outputs1,
             this_layer_outputs2 = last_layer_outputs2,
             this_layer_outputs3 = last_layer_outputs3,
             this_layer_outputs4 = last_layer_outputs4,
             this_layer_outputs5 = last_layer_outputs5,
             this_layer_outputs6 = last_layer_outputs6,
             this_layer_outputs7 = last_layer_outputs7,
             this_layer_outputs8 = last_layer_outputs8
         ) {
        
        // Get information about the layer:
        int ni = architecture[l];
        int no = architecture[l + 1];

        const float * layer_weights = w[l];
        int w_stride = w_strides[l];

        UpdateFloat * layer_updates = w_updates[l];
        UpdateFloat * layer_bias_updates  = b_updates[l];
        
        last_layer_outputs1 = this_layer_outputs1 - ni;
        last_layer_outputs2 = this_layer_outputs2 - ni;
        last_layer_outputs3 = this_layer_outputs3 - ni;
        last_layer_outputs4 = this_layer_outputs4 - ni;
        last_layer_outputs5 = this_layer_outputs5 - ni;
        last_layer_outputs6 = this_layer_outputs6 - ni;
        last_layer_outputs7 = this_layer_outputs7 - ni;
        last_layer_outputs8 = this_layer_outputs8 - ni;
        
        float prev_output1 = (tid >= no ? 0.0 : this_layer_outputs1[tid]);
        float prev_output2 = (tid >= no ? 0.0 : this_layer_outputs2[tid]);
        float prev_output3 = (tid >= no ? 0.0 : this_layer_outputs3[tid]);
        float prev_output4 = (tid >= no ? 0.0 : this_layer_outputs4[tid]);
        float prev_output5 = (tid >= no ? 0.0 : this_layer_outputs5[tid]);
        float prev_output6 = (tid >= no ? 0.0 : this_layer_outputs6[tid]);
        float prev_output7 = (tid >= no ? 0.0 : this_layer_outputs7[tid]);
        float prev_output8 = (tid >= no ? 0.0 : this_layer_outputs8[tid]);

        float error1 = scratch1[tid];
        float error2 = scratch2[tid];
        float error3 = scratch3[tid];
        float error4 = scratch4[tid];
        float error5 = scratch5[tid];
        float error6 = scratch6[tid];
        float error7 = scratch7[tid];
        float error8 = scratch8[tid];
        
        float d1 = (tid >= no ? 0.0 : delta(prev_output1, error1, activation));
        float d2 = (tid >= no ? 0.0 : delta(prev_output2, error2, activation));
        float d3 = (tid >= no ? 0.0 : delta(prev_output3, error3, activation));
        float d4 = (tid >= no ? 0.0 : delta(prev_output4, error4, activation));
        float d5 = (tid >= no ? 0.0 : delta(prev_output5, error5, activation));
        float d6 = (tid >= no ? 0.0 : delta(prev_output6, error6, activation));
        float d7 = (tid >= no ? 0.0 : delta(prev_output7, error7, activation));
        float d8 = (tid >= no ? 0.0 : delta(prev_output8, error8, activation));

        if (l > 0) {
            // Make sure all threads have caught up so that we can modify error
            // without affecting them
            __syncthreads();

            // Broadcast the d values so that we can use them to calculate the
            // errors
            scratch1[tid] = d1;
            scratch2[tid] = d2;
            scratch3[tid] = d3;
            scratch4[tid] = d4;
            scratch5[tid] = d5;
            scratch6[tid] = d6;
            scratch7[tid] = d7;
            scratch8[tid] = d8;

            // Make sure everything can get its d value
            __syncthreads();
            
            double total1 = 0.0, total2 = 0.0, total3 = 0.0, total4 = 0.0,
                total5 = 0.0, total6 = 0.0, total7 = 0.0, total8 = 0.0;
            if (tid < ni) {
                for (unsigned o = 0;  o < no;  ++o) {
                    float d1 = scratch1[o];
                    float d2 = scratch2[o];
                    float d3 = scratch3[o];
                    float d4 = scratch4[o];
                    float d5 = scratch5[o];
                    float d6 = scratch6[o];
                    float d7 = scratch7[o];
                    float d8 = scratch8[o];
                    
                    float w = layer_weights[tid * w_stride + o];

                    float update1 = d1 * w;
                    float update2 = d2 * w;
                    float update3 = d3 * w;
                    float update4 = d4 * w;
                    float update5 = d5 * w;
                    float update6 = d6 * w;
                    float update7 = d7 * w;
                    float update8 = d8 * w;

                    total1 += update1;
                    total2 += update2;
                    total3 += update3;
                    total4 += update4;
                    total5 += update5;
                    total6 += update6;
                    total7 += update7;
                    total8 += update8;
                }
            }

            // Wait for everything to finish so that we can overwrite the d
            // values with the new errors
            __syncthreads();
            
            scratch1[tid] = total1;
            scratch2[tid] = total2;
            scratch3[tid] = total3;
            scratch4[tid] = total4;
            scratch5[tid] = total5;
            scratch6[tid] = total6;
            scratch7[tid] = total7;
            scratch8[tid] = total8;
        }

        // Again, threads indexed too low just leave
        if (tid >= no) continue;

        /* Update the weights. */
        float k1 = example_weight1.x * learning_rate;
        float k2 = example_weight1.y * learning_rate;
        float k3 = example_weight1.z * learning_rate;
        float k4 = example_weight1.w * learning_rate;
        float k5 = example_weight2.x * learning_rate;
        float k6 = example_weight2.y * learning_rate;
        float k7 = example_weight2.z * learning_rate;
        float k8 = example_weight2.w * learning_rate;

        /* Now for the updates.  In order to avoid trying to write the same
           memory over and over, we stagger the starting points so that
           each example will start at a different place, thus minimising
           conflicting writes when we have multiple multiprocessors working
           on the same thing. */

        int thread_stride = ni / num_threads_in_block;
        if (thread_stride == 0) thread_stride = 1;

        int start_at = (block_num * thread_stride) % ni;

        for (unsigned i_ = start_at;  i_ < ni + start_at;  ++i_) {

            // Get the real index of i
            unsigned i = i_ - (i_ >= ni) * ni;

            float prev1 = (l == 0 ? input1[i] : last_layer_outputs1[i]); 
            float prev2 = (l == 0 ? input2[i] : last_layer_outputs2[i]); 
            float prev3 = (l == 0 ? input3[i] : last_layer_outputs3[i]); 
            float prev4 = (l == 0 ? input4[i] : last_layer_outputs4[i]); 
            float prev5 = (l == 0 ? input5[i] : last_layer_outputs5[i]); 
            float prev6 = (l == 0 ? input6[i] : last_layer_outputs6[i]); 
            float prev7 = (l == 0 ? input7[i] : last_layer_outputs7[i]); 
            float prev8 = (l == 0 ? input8[i] : last_layer_outputs8[i]); 

            float update1 = k1 * d1 * prev1;
            float update2 = k2 * d2 * prev2;
            float update3 = k3 * d3 * prev3;
            float update4 = k4 * d4 * prev4;
            float update5 = k5 * d5 * prev5;
            float update6 = k6 * d6 * prev6;
            float update7 = k7 * d7 * prev7;
            float update8 = k8 * d8 * prev8;

            float update
                = update1 + update2 + update3 + update4
                + update5 + update6 + update7 + update8;

#if defined(__DEVICE_EMULATION__)
            //__syncthreads();

            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev1, k1, d1, update1, (float)layer_updates[i * w_stride + tid]);
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev2, k2, d2, update2, (float)layer_updates[i * w_stride + tid]);
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev3, k3, d3, update3, (float)layer_updates[i * w_stride + tid]);
                fprintf(stderr, "update for layer 2 i=0 o=%d = %.15g * %.15g * %.15g = %.15g before update %.15g\n", tid, prev4, k4, d4, update4, (float)layer_updates[i * w_stride + tid]);
            }
#endif

            atomic_add(layer_updates[i * w_stride + tid], update);

#if defined(__DEVICE_EMULATION__)
            if (tid < 10 && block_num == 0 && l == 2 && i == 0) {
                fprintf(stderr, "                          after %.15g\n", (float)layer_updates[i * w_stride + tid]);
            }
#endif

        }

        /* Update the bias */
        double update
            = double(k1 * d1)
            + double(k2 * d2)
            + double(k3 * d3)
            + double(k4 * d4)
            + double(k5 * d5)
            + double(k6 * d6)
            + double(k7 * d7)
            + double(k8 * d8)
            ;

        atomic_add(layer_bias_updates[tid], update);
    }
}
#endif

__global__ void
train_examples_kernel(const float * feature_vectors,  // feature vector [ni]
                      int feature_vector_width,
                      const int * labels,
                      const float * example_weights,
                      int num_layers,
                      const float * w,  // weights for each layer
                      const float * biases, // for each layer
                      const int * architecture,
                      const int * w_strides,
                      UpdateFloat * const * w_updates, // wt updates for each layer
                      UpdateFloat * const * b_updates, // bias upd for each layer
                      int activation,            // activation function
                      float fire,   // target value for firing neuron
                      float inhibit, // target value for inhibited neuron)
                      float learning_rate,
                      int num_threads_in_block,
                      int num_threads_on_multiprocessor,
                      int total_neurons,
                      float * layer_outputs,  // scratch space[total neurons]
                      int examples_per_block,
                      int total_num_examples,
                      int max_width)
{
    const unsigned block_num  = blockIdx.x;
    
    /* Where we accumulate our errors, layer by layer.  The size is that of
       the largest dimension. */
    extern __shared__ float scratch[];
    
    /* The layer outputs (activation of the neurons).  This is where the
       shared memory goes to.  Note that we store only the activated outputs,
       not the inputs.

       blockDim.x gives us the number of threads, which is also the size of
       the errors array, so that our layer outputs have to start at this
       offset.
    */

    // Get our private scratch memory for this block
    layer_outputs += block_num * total_neurons * 4;
    
    unsigned example_num_base = block_num * examples_per_block;
    unsigned last_example = min(total_num_examples, example_num_base + examples_per_block);

    unsigned example_num = example_num_base;

#if 0
    for (;  example_num < last_example - 7;  example_num += 8) {
        const float * input1 = feature_vectors + example_num * feature_vector_width;
        const float * input2 = input1 + feature_vector_width;
        const float * input3 = input2 + feature_vector_width;
        const float * input4 = input3 + feature_vector_width;
        const float * input5 = input4 + feature_vector_width;
        const float * input6 = input5 + feature_vector_width;
        const float * input7 = input6 + feature_vector_width;
        const float * input8 = input7 + feature_vector_width;

        int4 label1, label2;
        label1.x = labels[example_num];
        label1.y = labels[example_num + 1];
        label1.z = labels[example_num + 2];
        label1.w = labels[example_num + 3];
        label2.x = labels[example_num + 4];
        label2.y = labels[example_num + 5];
        label2.z = labels[example_num + 6];
        label2.w = labels[example_num + 7];

        float4 example_weight1, example_weight2;
        example_weight1.x = example_weights[example_num];
        example_weight1.y = example_weights[example_num + 1];
        example_weight1.z = example_weights[example_num + 2];
        example_weight1.w = example_weights[example_num + 3];
        example_weight2.x = example_weights[example_num + 4];
        example_weight2.y = example_weights[example_num + 5];
        example_weight2.z = example_weights[example_num + 6];
        example_weight2.w = example_weights[example_num + 7];

        float * scratch1 = scratch;
        float * scratch2 = scratch1 + max_width;
        float * scratch3 = scratch2 + max_width;
        float * scratch4 = scratch3 + max_width;
        float * scratch5 = scratch4 + max_width;
        float * scratch6 = scratch5 + max_width;
        float * scratch7 = scratch6 + max_width;
        float * scratch8 = scratch7 + max_width;

        float * layer_outputs1 = layer_outputs;
        float * layer_outputs2 = layer_outputs1 + total_neurons;
        float * layer_outputs3 = layer_outputs2 + total_neurons;
        float * layer_outputs4 = layer_outputs3 + total_neurons;
        float * layer_outputs5 = layer_outputs4 + total_neurons;
        float * layer_outputs6 = layer_outputs5 + total_neurons;
        float * layer_outputs7 = layer_outputs6 + total_neurons;
        float * layer_outputs8 = layer_outputs7 + total_neurons;
        
        train_8_examples(input1, input2, input3, input4,
                         input5, input6, input7, input8,
                         label1, label2, example_weight1, example_weight2,
                         num_layers,
                         scratch1, scratch2, scratch3, scratch4,
                         scratch5, scratch6, scratch7, scratch8,
                         w, biases, architecture, w_strides,
                         w_updates, b_updates,
                         activation, fire, inhibit, learning_rate,
                         num_threads_in_block, total_neurons,
                         layer_outputs1, layer_outputs2, layer_outputs3,
                         layer_outputs4, layer_outputs5, layer_outputs6,
                         layer_outputs7, layer_outputs8);

    }
#endif

#if 0
    for (;  example_num < last_example - 3;  example_num += 4) {
        const float * input1 = feature_vectors + example_num * feature_vector_width;
        const float * input2 = input1 + feature_vector_width;
        const float * input3 = input2 + feature_vector_width;
        const float * input4 = input3 + feature_vector_width;

        int4 label;
        label.x = labels[example_num];
        label.y = labels[example_num + 1];
        label.z = labels[example_num + 2];
        label.w = labels[example_num + 3];

        float4 example_weight;
        example_weight.x = example_weights[example_num];
        example_weight.y = example_weights[example_num + 1];
        example_weight.z = example_weights[example_num + 2];
        example_weight.w = example_weights[example_num + 3];

        float * scratch1 = scratch;
        float * scratch2 = scratch1 + max_width;
        float * scratch3 = scratch2 + max_width;
        float * scratch4 = scratch3 + max_width;

        float * layer_outputs1 = layer_outputs;
        float * layer_outputs2 = layer_outputs1 + total_neurons;
        float * layer_outputs3 = layer_outputs2 + total_neurons;
        float * layer_outputs4 = layer_outputs3 + total_neurons;
        
        train_4_examples(input1, input2, input3, input4,
                         label, example_weight,
                         num_layers, scratch1, scratch2, scratch3, scratch4,
                         w, biases, architecture, w_strides,
                         w_updates, b_updates,
                         activation, fire, inhibit, learning_rate,
                         num_threads_in_block, total_neurons,
                         layer_outputs1, layer_outputs2, layer_outputs3,
                         layer_outputs4);

    }
#endif

    // Do any others singly
    for (;  example_num < last_example;  ++example_num) {

        const float * input = feature_vectors + example_num * feature_vector_width;

        int label = labels[example_num];

        float example_weight = example_weights[example_num];

        train_example(input, label, example_weight,
                      num_layers, scratch, w, biases, architecture, w_strides,
                      w_updates, b_updates,
                      activation, fire, inhibit, learning_rate,
                      num_threads_in_block,
                      num_threads_on_multiprocessor,
                      total_neurons, layer_outputs);
    }
}


namespace ML {
namespace CUDA {

struct Backprop::Plan {
    int num_layers;

    vector<int> architecture;
    DeviceData<int> d_architecture;

    DeviceData<float> d_weights;

    DeviceData<float> d_biases;

    vector<int> w_strides;
    DeviceData<int> d_w_strides;

    Activation activation;
    float fire;
    float inhibit;
    float learning_rate;

    int max_width;
    int total_neurons;

    // We need our grid size to be exactly the maximum width of the output
    dim3 threads;
    
    size_t shared_mem_size;

    Plan(int num_layers,
         const int * architecture,
         const float * const * weights,
         const float * const * biases,
         const int * w_strides,
         Activation activation,
         float fire,
         float inhibit,
         float learning_rate,
         bool on_host)
        : num_layers(num_layers),
          architecture(architecture, architecture + num_layers + 1),
          w_strides(w_strides, w_strides + num_layers),
          activation(activation),
          fire(fire),
          inhibit(inhibit),
          learning_rate(learning_rate)
    {
        //cerr << "plan: num_layers = " << num_layers << endl;

        d_architecture.init(architecture, num_layers + 1);

        size_t total_weights_size = 0;
        size_t total_bias_size = 0;

        for (unsigned l = 0;  l < num_layers;  ++l) {
            int ni = architecture[l];
            int no = architecture[l + 1];
            int w_stride = w_strides[l];
            total_weights_size += ni * w_stride;
            total_bias_size += no;
            // TODO: align?
        }

        d_weights.init(total_weights_size);
        d_biases.init(total_bias_size);
        
        // Now copy them all in

        size_t weights_start_offset = 0;
        size_t bias_start_offset = 0;
        
        for (unsigned l = 0;  l < num_layers;  ++l) {
            int ni = architecture[l];
            int no = architecture[l + 1];
            int w_stride = w_strides[l];
            size_t w_size = ni * w_stride;

            hipError_t err
                = hipMemcpy(d_weights + weights_start_offset,
                             weights[l],
                             w_size * sizeof(float),
                             hipMemcpyHostToDevice);
            
            if (err != hipSuccess)
                throw Exception(hipGetErrorString(err));

            err = hipMemcpy(d_biases + bias_start_offset,
                             biases[l],
                             no * sizeof(float),
                             hipMemcpyHostToDevice);
            
            if (err != hipSuccess)
                throw Exception(hipGetErrorString(err));
            
            weights_start_offset += ni * w_stride;
            bias_start_offset += no;
            // TODO: align?
        }

        d_w_strides.init(w_strides, num_layers);
        
        max_width = 0;
        total_neurons = 0;

        for (unsigned l = 0;  l <= num_layers;  ++l) {
            max_width = max(max_width, architecture[l]);
            total_neurons += architecture[l];
        }

        // We need our grid size to be exactly the maximum width of the output
        threads = dim3(max_width);

        // Since we do 4 examples per loop, we need enough memory for all of
        // the four outputs for a single layer
        shared_mem_size = max_width * sizeof(float)* 4;
    }
};

struct Backprop::Context {

    const Plan & plan;
    
    DeviceData<float> d_feature_vectors;
    DeviceData<float> d_example_weights;
    DeviceData<int> d_labels;
        
    float * const * weight_updates;
    float * const * bias_updates;

    vector<DeviceData<UpdateFloat> > d_weight_updates_storage;
    vector<UpdateFloat *> weight_updates_vec;
    DeviceData<UpdateFloat *> d_weight_updates;
    
    vector<DeviceData<UpdateFloat> > d_bias_updates_storage;
    vector<UpdateFloat *> bias_updates_vec;
    DeviceData<UpdateFloat *> d_bias_updates;

    DeviceData<float> d_layer_outputs;

    dim3 grid;

    int num_feature_vectors;
    int feature_vector_width;
    int num_examples_per_invocation;

    Context(const Plan & plan,
            const float * feature_vectors,
            int num_feature_vectors,
            const float * example_weights,
            const int * labels,
            float * const * weight_updates,
            float * const * bias_updates,
            float & correct,
            float & total,
            float & rms_error)
        : plan(plan), weight_updates(weight_updates),
          bias_updates(bias_updates), num_feature_vectors(num_feature_vectors),
          feature_vector_width(feature_vector_width)
    {
        feature_vector_width = plan.architecture[0];
        
        //cerr << "num_feature_vectors = " << num_feature_vectors << endl;
        //cerr << "feature_vector_width = " << feature_vector_width
        //     << endl;

        d_feature_vectors.init(feature_vectors,
                               num_feature_vectors * feature_vector_width);
        
        d_example_weights.init(example_weights, num_feature_vectors);
        
        d_labels.init(labels, num_feature_vectors);
        
        d_weight_updates_storage.resize(plan.num_layers);
        weight_updates_vec.resize(plan.num_layers);
        
        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            d_weight_updates_storage[l].init_zeroed(ni * w_stride);
            weight_updates_vec[l] = d_weight_updates_storage[l];
        }

        d_weight_updates.init(&weight_updates_vec[0], plan.num_layers);

        d_bias_updates_storage.resize(plan.num_layers);
        bias_updates_vec.resize(plan.num_layers);

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];
            d_bias_updates_storage[l].init_zeroed(no);
            bias_updates_vec[l] = d_bias_updates_storage[l];
        }

        d_bias_updates.init(&bias_updates_vec[0], plan.num_layers);

        num_examples_per_invocation = 4;//16;

        int grid_size = rudiv(num_feature_vectors, num_examples_per_invocation);

        // Get the scratch space
        d_layer_outputs.init(plan.total_neurons * grid_size * 4);
        
        // Our grid size is one per example
        grid = dim3(grid_size);
    }

    void execute()
    {
        train_examples_kernel<<<grid, plan.threads, plan.shared_mem_size>>>
            (d_feature_vectors,
             feature_vector_width,
             d_labels,
             d_example_weights,
             plan.num_layers,
             plan.d_weights,
             plan.d_biases,
             plan.d_architecture,
             plan.d_w_strides,
             d_weight_updates,
             d_bias_updates,
             plan.activation,
             plan.fire,
             plan.inhibit,
             plan.learning_rate,
             grid.x,
             plan.threads.x,
             plan.total_neurons,
             d_layer_outputs,
             num_examples_per_invocation,
             num_feature_vectors /* total num examples */,
             plan.max_width);

        //cerr << "launched" << endl;
    }
    
    void synchronize()
    {
        //cerr << "waiting for execution" << endl;
        hipError_t err = hipDeviceSynchronize();
        
        if (err != hipSuccess)
            throw Exception(hipGetErrorString(err));

        //cerr << "copying memory back" << endl;

        


        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int ni = plan.architecture[l];
            int w_stride = plan.w_strides[l];
            
            UpdateFloat sync_to[ni * w_stride];

            d_weight_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + ni * w_stride, weight_updates[l]);

#if 0
            cerr << "first 10 weight updates for layer " << l << ": ";
            for (unsigned i = 0;  i < 10;  ++i)
                cerr << sync_to[i] << " ";
            cerr << endl;
#endif
        }

        for (unsigned l = 0;  l < plan.num_layers;  ++l) {
            int no = plan.architecture[l + 1];

            UpdateFloat sync_to[no];
            
            d_bias_updates_storage[l].sync(sync_to);
            std::copy(sync_to, sync_to + no, bias_updates[l]);

#if 0            
            cerr << "first 10 bias updates for layer " << l << ": ";
            for (unsigned i = 0;  i < 10;  ++i)
                cerr << sync_to[i] << " ";
            cerr << endl;
#endif
        }
    }
};

boost::shared_ptr<Backprop::Plan>
Backprop::
plan(int num_layers,
     const int * architecture,
     const float * const * weights,
     const float * const * biases,
     const int * w_strides,
     Activation activation,
     float fire,
     float inhibit,
     float learning_rate,
     bool on_host) const
{
    boost::shared_ptr<Plan> result
        (new Plan(num_layers, architecture, weights, biases, w_strides,
                  activation, fire, inhibit, learning_rate, on_host));

    return result;
}

boost::shared_ptr<Backprop::Context>
Backprop::
execute(const Plan & plan,
        const float * feature_vectors,
        int num_feature_vectors,
        const float * example_weights,
        const int * labels,
        float * const * weight_updates,
        float * const * bias_updates,
        float & correct,
        float & total,
        float & rms_error) const
{
    boost::shared_ptr<Context> result
        (new Context(plan, feature_vectors, num_feature_vectors,
                     example_weights, labels,
                     weight_updates, bias_updates,
                     correct, total, rms_error));

    result->execute();

    return result;
}

/** Wait for the given context to be finished. */
void
Backprop::
synchronize(Context & context) const
{
    context.synchronize();
}


} // namespace CUDA
} // namespace ML
